#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <stdio.h>


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
#endif

extern "C"
void runCudaPart();

__global__ void helloCUDA(glm::vec3 v)
{
    int tid = blockIdx.x;
    printf("Hello block %d thread %d, x=%f\n",tid , threadIdx.x, v.x);
}

void runCudaPart()
{
    // all your cuda code here
    glm::vec3 v(0.1f, 0.2f, 0.3f);
//    helloCUDA<<<1, 5>>>(v); // 1 block, 5 GPU threads
    helloCUDA<<<5,1>>>(v); // 5 blocks, 1 GPU thread each
    hipDeviceSynchronize();
}
