#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   snow.cu
**   Author: mliberma
**   Created: 7 Apr 2014
**
**************************************************************************/

#ifndef SNOW_CU
#define SNOW_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <glm/geometric.hpp>

#define CUDA_INCLUDE
#include "sim/particle.h"
#include "cuda/functions.h"

void registerVBO( hipGraphicsResource **resource, GLuint vbo )
{
    checkCudaErrors( hipGraphicsGLRegisterBuffer(resource, vbo, cudaGraphicsMapFlagsWriteDiscard) );
}

void unregisterVBO( hipGraphicsResource *resource )
{
    checkCudaErrors( hipGraphicsUnregisterResource(resource) );
}

__global__ void snow_kernel( float time, Particle *particles )
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    glm::vec3 pn = glm::normalize( particles[index].position );
    particles[index].position += 0.05f*sinf(10*time)*pn;
}

void updateParticles( hipGraphicsResource **resource, float time, int particleCount )
{

    checkCudaErrors( hipGraphicsMapResources(1, resource, 0) );
    Particle *particles;
    size_t size;
    checkCudaErrors( hipGraphicsResourceGetMappedPointer((void**)&particles, &size, *resource) );
    snow_kernel<<< particleCount/512, 512 >>>( time, particles );
    checkCudaErrors( hipGraphicsUnmapResources(1, resource, 0) );
}

#endif // SNOW_CU
