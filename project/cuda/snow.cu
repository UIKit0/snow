#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   snow.cu
**   Author: mliberma
**   Created: 7 Apr 2014
**
**************************************************************************/

#ifndef SNOW_CU
#define SNOW_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <glm/geometric.hpp>

#define CUDA_INCLUDE
#include "sim/particle.h"
#include "cuda/functions.h"

void registerVBO( hipGraphicsResource **resource, GLuint vbo )
{
    checkCudaErrors( hipGraphicsGLRegisterBuffer(resource, vbo, cudaGraphicsMapFlagsWriteDiscard) );
}

void unregisterVBO( hipGraphicsResource *resource )
{
    checkCudaErrors( hipGraphicsUnregisterResource(resource) );
}

__global__ void snow_kernel( float time, Particle *particles )
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    vec3 pn = vec3::normalize( particles[index].position );
    particles[index].position += 0.05f*sinf(6*time)*pn;
}

void updateParticles( hipGraphicsResource **resource, float time, int particleCount )
{

    hipGraphicsMapResources( 1, resource, 0 );
    Particle *particles;
    size_t size;
    hipGraphicsResourceGetMappedPointer( (void**)&particles, &size, *resource );
    snow_kernel<<< particleCount/512, 512 >>>( time, particles );
    checkCudaErrors( hipDeviceSynchronize() );
    hipGraphicsUnmapResources( 1, resource, 0 );
}

#endif // SNOW_CU
