#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   vector.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 15 Apr 2014
**
**************************************************************************/

#ifndef VECTOR_CU
#define VECTOR_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common/math.h"

#define GLM_FORCE_RADIANS
#include "glm/vec3.hpp"

struct vec3
{
    union {
        float data[3];
        struct { float x, y, z; };
    };

    __host__ __device__ __forceinline__
    vec3() { x = 0.f; y = 0.f; z = 0.f; }

    __host__ __device__ __forceinline__
    vec3( float a, float b, float c ) { x = a; y = b; z = c; }

    __host__ __device__ __forceinline__
    vec3( const vec3 &v ) { x = v.x; y = v.y; z = v.z; }

    __host__ __device__ __forceinline__
    vec3& operator = ( const vec3 &rhs ) { x = rhs.x; y = rhs.y; z = rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3( const glm::vec3 &v ) { x = v.x; y = v.y; z = v.z; }

    __host__ __device__ __forceinline__
    vec3& operator = ( const glm::vec3 &rhs ) { x = rhs.x; y = rhs.y; z = rhs.z; return *this; }

    __host__ __device__ __forceinline__
    glm::vec3 toGLM() const { return glm::vec3( x, y, z ); }

    __host__ __device__ __forceinline__
    static float dot( const vec3 &a, const vec3 &b ) { return a.x*b.x + a.y*b.y + a.z*b.z; }

    __host__ __device__ __forceinline__
    static vec3 cross( const vec3 &a, const vec3 &b )
    {
        return vec3( a.y*b.z - a.z*b.y,
                     a.z*b.x - a.x*b.z,
                     a.x*b.y - a.y*b.x );
    }

    __host__ __device__ __forceinline__
    static vec3 floor( const vec3 &v ) { return vec3( floorf(v.x), floorf(v.y), floorf(v.z) ); }

    __host__ __device__ __forceinline__
    static vec3 ceil( const vec3 &v ) { return vec3( ceilf(v.x), ceilf(v.y), ceilf(v.z) ); }

    __host__ __device__ __forceinline__
    static float length2( const vec3 &v ) { return v.x*v.x + v.y*v.y + v.z*v.z; }

    __host__ __device__ __forceinline__
    static float length( const vec3 &v ) { return sqrtf( v.x*v.x + v.y*v.y + v.z*v.z ); }

    __host__ __device__ __forceinline__
    static vec3 normalize( const vec3 &v ) { float f = 1.f/sqrtf(v.x*v.x+v.y*v.y+v.z*v.z); return vec3( f*v.x, f*v.y, f*v.z ); }

    __host__ __device__ __forceinline__
    vec3& operator += ( const vec3 &rhs ) { x += rhs.x; y += rhs.y; z += rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator + ( const vec3 &rhs ) const { return vec3( x+rhs.x, y+rhs.y, z+rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator -= ( const vec3 &rhs ) { x -= rhs.x; y -= rhs.y; z -= rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator - ( const vec3 &rhs ) const { return vec3( x-rhs.x, y-rhs.y, z-rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator *= ( const vec3 &rhs ) { x *= rhs.x; y *= rhs.y; z *= rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator * ( const vec3 &rhs ) { return vec3( x*rhs.x, y*rhs.y, z*rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator /= ( const vec3 &rhs ) { x /= rhs.x; y /= rhs.y; z /= rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator / ( const vec3 &rhs ) const { return vec3( x/rhs.x, y/rhs.y, z/rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator *= ( float f ) { x *= f; y *= f; z *= f; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator * ( float f ) { return vec3( f*x, f*y, f*z ); }

    __host__ __device__ __forceinline__
    vec3& operator /= ( float f ) { float fi = 1./f; x *= fi; y *= fi; z *= fi; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator / ( float f ) { float fi = 1.f/f; return vec3( x*fi, y*fi, z*fi ); }

};

__host__ __device__ __forceinline__
vec3 operator - ( const vec3 &v ) { return vec3( -v.x, -v.y, -v.z ); }

__host__ __device__ __forceinline__
vec3 operator * ( float f, const vec3 &v ) { return vec3( f*v.x, f*v.y, f*v.z ); }

#endif // VECTOR_CU
