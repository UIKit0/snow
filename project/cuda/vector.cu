#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   vector.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 15 Apr 2014
**
**************************************************************************/

#ifndef VECTOR_CU
#define VECTOR_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common/math.h"




struct vec3
{
    union {
        float data[3];
        struct { float x, y, z; };
    };

    __host__ __device__ __forceinline__
    vec3() { x = 0.f; y = 0.f; z = 0.f; }

    __host__ __device__ __forceinline__
    vec3( float v ) { x = v; y = v; z = v; }

    __host__ __device__ __forceinline__
    vec3( float xx, float yy, float zz ) { x = xx; y = yy; z = zz; }

    __host__ __device__ __forceinline__
    vec3( const vec3 &v ) { x = v.x; y = v.y; z = v.z; }

    __host__ __device__ __forceinline__
    vec3& operator = ( const vec3 &rhs ) { x = rhs.x; y = rhs.y; z = rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3( const glm::vec3 &v ) { x = v.x; y = v.y; z = v.z; }

    __host__ __device__ __forceinline__
    vec3& operator = ( const glm::vec3 &rhs ) { x = rhs.x; y = rhs.y; z = rhs.z; return *this; }

    __host__ __device__ __forceinline__
    float& operator [] ( int i ) { return data[i]; }

    __host__ __device__ __forceinline__
    float operator [] ( int i ) const { return data[i]; }

    __host__ __device__ __forceinline__
    glm::vec3 toGLM() const { return glm::vec3( x, y, z ); }

    __host__ __device__ __forceinline__
    static float dot( const vec3 &a, const vec3 &b ) { return a.x*b.x + a.y*b.y + a.z*b.z; }

    __host__ __device__ __forceinline__
    static vec3 cross( const vec3 &a, const vec3 &b )
    {
        return vec3( a.y*b.z - a.z*b.y,
                     a.z*b.x - a.x*b.z,
                     a.x*b.y - a.y*b.x );
    }

    __host__ __device__ __forceinline__
    static vec3 floor( const vec3 &v ) { return vec3( floorf(v.x), floorf(v.y), floorf(v.z) ); }

    __host__ __device__ __forceinline__
    static vec3 ceil( const vec3 &v ) { return vec3( ceilf(v.x), ceilf(v.y), ceilf(v.z) ); }

    __host__ __device__ __forceinline__
    static vec3 abs( const vec3 &v ) { return vec3( fabs(v.x), fabs(v.y), fabs(v.z) ); }

    __host__ __device__ __forceinline__
    static vec3 round( const vec3 &v ) { return vec3( roundf(v.x), roundf(v.y), roundf(v.z) ); }

    //From http://stackoverflow.com/questions/1903954/is-there-a-standard-sign-function-signum-sgn-in-c-c
    __host__ __device__ __forceinline__
    static float sign( const float v ) { return (0 < v) - (v < 0);}

    //From http://stackoverflow.com/questions/1903954/is-there-a-standard-sign-function-signum-sgn-in-c-c
    __host__ __device__ __forceinline__
    static vec3 sign( const vec3 &v ) { return vec3(sign(v.x), sign(v.y), sign(v.z) );}

    __host__ __device__ __forceinline__
    static float length2( const vec3 &v ) { return v.x*v.x + v.y*v.y + v.z*v.z; }

    __host__ __device__ __forceinline__
    static float length( const vec3 &v ) { return sqrtf( v.x*v.x + v.y*v.y + v.z*v.z ); }

    __host__ __device__ __forceinline__
    static vec3 normalize( const vec3 &v ) { float f = 1.f/sqrtf(v.x*v.x+v.y*v.y+v.z*v.z); return vec3( f*v.x, f*v.y, f*v.z ); }

    __host__ __device__ __forceinline__
    vec3& operator += ( const vec3 &rhs ) { x += rhs.x; y += rhs.y; z += rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator + ( const vec3 &rhs ) const { return vec3( x+rhs.x, y+rhs.y, z+rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator -= ( const vec3 &rhs ) { x -= rhs.x; y -= rhs.y; z -= rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator - ( const vec3 &rhs ) const { return vec3( x-rhs.x, y-rhs.y, z-rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator *= ( const vec3 &rhs ) { x *= rhs.x; y *= rhs.y; z *= rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator * ( const vec3 &rhs ) const { return vec3( x*rhs.x, y*rhs.y, z*rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator /= ( const vec3 &rhs ) { x /= rhs.x; y /= rhs.y; z /= rhs.z; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator / ( const vec3 &rhs ) const { return vec3( x/rhs.x, y/rhs.y, z/rhs.z ); }

    __host__ __device__ __forceinline__
    vec3& operator *= ( float f )  { x *= f; y *= f; z *= f; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator * ( float f ) const { return vec3( f*x, f*y, f*z ); }

    __host__ __device__ __forceinline__
    vec3& operator /= ( float f ) { float fi = 1./f; x *= fi; y *= fi; z *= fi; return *this; }

    __host__ __device__ __forceinline__
    vec3 operator / ( float f ) const { float fi = 1.f/f; return vec3( x*fi, y*fi, z*fi ); }



};

__host__ __device__ __forceinline__
vec3 operator - ( const vec3 &v ) { return vec3( -v.x, -v.y, -v.z ); }

__host__ __device__ __forceinline__
vec3 operator * ( float f, const vec3 &v ) { return vec3( f*v.x, f*v.y, f*v.z ); }

#endif // VECTOR_CU
