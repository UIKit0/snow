#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   max.cu
**   Author: mliberma
**   Created: 8 Apr 2014
**
**************************************************************************/

#ifndef MAX_CU
#define MAX_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include "glm/common.hpp"
#include "glm/geometric.hpp"

#define CUDA_INCLUDE
#include "common/common.h"
#include "common/math.h"
#include "cuda/functions.h"
#include "geometry/bbox.h"
#include "geometry/mesh.h"
#include "sim/particle.h"
#include "sim/grid.h"

#endif // MAX_CU
