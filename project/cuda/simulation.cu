#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   simulation.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 17 Apr 2014
**
**************************************************************************/

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define CUDA_INCLUDE
#include "sim/collider.h"
#include "sim/material.h"
#include "sim/parameters.h"
#include "sim/particle.h"
#include "sim/particlegrid.h"

#include "common/math.h"

#include "cuda/collider.cu"
#include "cuda/decomposition.cu"
#include "cuda/weighting.cu"

#include "cuda/functions.h"

__host__ __device__ __forceinline__
bool withinBoundsInclusive( const float &v, const float &min, const float &max )
{
    return ( v >= min && v <= max );
}

__host__ __device__ __forceinline__
bool withinBoundsInclusive( const glm::ivec3 &v, const glm::ivec3 &min, const glm::ivec3 &max )
{
    return withinBoundsInclusive(v.x, min.x, max.x) && withinBoundsInclusive(v.y, min.y, max.y) && withinBoundsInclusive(v.z, min.z, max.z);
}


__host__ __device__ __forceinline__
void gridIndexToIJK( int idx, int &i, int &j, int &k,const  glm::ivec3 &nodeDim )
{
    i = idx / (nodeDim.y*nodeDim.z);
    idx = idx % (nodeDim.y*nodeDim.z);
    j = idx / nodeDim.z;
    k = idx % nodeDim.z;
}

__host__ __device__  __forceinline__
int getGridIndex( int i, int j, int k, const glm::ivec3 &nodeDim)
{
    return (i*(nodeDim.y*nodeDim.z) + j*(nodeDim.z) + k);
}

__host__ __device__ __forceinline__
void gridIndexToIJK( int idx, const  glm::ivec3 &nodeDim, glm::ivec3 &IJK )
{
    gridIndexToIJK(idx, IJK.x, IJK.y, IJK.z, nodeDim);
}

__host__ __device__ __forceinline__
int getGridIndex( const glm::ivec3 &IJK, const glm::ivec3 &nodeDim )
{
    return getGridIndex(IJK.x, IJK.y, IJK.z, nodeDim);
}

__device__ void computeSigma( Particle &particle, MaterialConstants *material, mat3 &sigma )
{
    mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re;
    computePD(Fe, Re);

    float muFp = material->mu*__expf(material->xi*(1-Jpp));
    float lambdaFp = material->lambda*__expf(material->xi*(1-Jpp));

//    sigma = (2*muFp*(Fe-Re)*mat3::transpose(Fe)+lambdaFp*(Jep-1)*Jep*mat3(1.0f)) * (particle.volume);
    sigma = (2*muFp*mat3::multiplyABt(Fe-Re, Fe) + mat3(lambdaFp*(Jep-1)*Jep)) * particle.volume;
}

__global__ void computeParticleGridTempData ( Particle *particleData, Grid *grid, MaterialConstants *material, ParticleGridTempData *particleGridTempData )
{
    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particleData[particleIdx];
    ParticleGridTempData &pgtd = particleGridTempData[particleIdx];

    pgtd.particleGridPos = (particle.position - grid->pos)/grid->h;
    computeSigma(particle, material, pgtd.sigma);
}

__device__ __forceinline__
void atomicAdd( vec3 *add, vec3 toAdd )
{
    atomicAdd(&(add->x), toAdd.x);
    atomicAdd(&(add->y), toAdd.y);
    atomicAdd(&(add->z), toAdd.z);
}

/**
 * Called on each particle.
 *
 * Each particle adds it's mass, velocity and force contribution to the grid nodes within 2h of itself.
 *
 * In:
 * particleData -- list of particles
 * grid -- Stores grid paramters
 * worldParams -- Global parameters dealing with the physics of the world
 *
 * Out:
 * nodes -- list of every node in grid ((dim.x+1)*(dim.y+1)*(dim.z+1))
 *
 */
__global__ void computeCellMassVelocityAndForceFast( Particle *particleData, Grid *grid, ParticleGridTempData *particleGridTempData, ParticleGrid::Node *nodes )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particleData[particleIdx];
    ParticleGridTempData &pgtd = particleGridTempData[particleIdx];

    glm::ivec3 currIJK;
    gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    currIJK.x += (int) pgtd.particleGridPos.x - 1; currIJK.y += (int) pgtd.particleGridPos.y - 1; currIJK.z += (int) pgtd.particleGridPos.z - 1;

    if (withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim)){
        ParticleGrid::Node &node = nodes[getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient(pgtd.particleGridPos-nodePosition, w, wg);

        atomicAdd(&node.mass, particle.mass*w);
        atomicAdd(&node.velocity, particle.velocity*particle.mass*w );
        atomicAdd(&node.force, pgtd.sigma*wg);
     }
}

/**
 * Called on each grid node.
 *
 * Updates the velocities of each grid node based on forces and collisions
 *
 * In:
 * nodes -- list of all nodes in the grid.
 * dt -- delta time, time step of simulation
 * colliders -- array of colliders in the scene.
 * numColliders -- number of colliders in the scene
 * worldParams -- Global parameters dealing with the physics of the world
 * grid -- parameters defining the grid
 *
 * Out:
 * nodes -- updated velocity and velocityChange
 *
 */
__global__ void updateNodeVelocities( ParticleGrid::Node *nodes, float dt, ImplicitCollider* colliders, int numColliders, MaterialConstants *material, Grid *grid, const vec3 gravity )
{
    int nodeIdx = blockIdx.x*blockDim.x + threadIdx.x;
    int gridI, gridJ, gridK;
    gridIndexToIJK(nodeIdx, gridI, gridJ, gridK, grid->dim+1);
    ParticleGrid::Node &node = nodes[nodeIdx];
    vec3 nodePosition = vec3(gridI, gridJ, gridK)*grid->h + grid->pos;

    float scale = ( node.mass > 1e-8 ) ? 1.f/node.mass : 0.f;

    node.velocity *= scale; //Have to normalize velocity by mass to conserve momentum

    // Update velocity with node force and gravity
    vec3 tmpVelocity = node.velocity + dt * ( node.force*scale + gravity );

    checkForAndHandleCollisions( colliders, numColliders, material->coeffFriction, nodePosition, tmpVelocity );
    node.velocityChange = tmpVelocity - node.velocity;
    node.velocity = tmpVelocity;
}

#define VEC2IVEC( V ) ( glm::ivec3((int)V.x, (int)V.y, (int)V.z) )

#define CLAMP( X, A, B ) ( (X < A) ? A : ((X > B) ? B : X) )

// Use weighting functions to compute particle velocity gradient and update particle velocity
__device__ void processGridVelocities( Particle &particle, Grid *grid, const ParticleGrid::Node *nodes, mat3 &velocityGradient, float alpha )
{
    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    // Compute neighborhood of particle in grid
    vec3 gridIndex = (pos - grid->pos) / h,
         gridMax = vec3::floor( gridIndex + vec3(2,2,2) ),
         gridMin = vec3::ceil( gridIndex - vec3(2,2,2) );
    glm::ivec3 maxIndex = glm::clamp( VEC2IVEC(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( VEC2IVEC(gridMin), glm::ivec3(0,0,0), dim );

    // For computing particle velocity gradient:
    //      grad(v_p) = sum( v_i * transpose(grad(w_ip)) ) = [3x3 matrix]
    // For updating particle velocity:
    //      v_PIC = sum( v_i * w_ip )
    //      v_FLIP = v_p + sum( dv_i * w_ip )
    //      v = (1-alpha)*v_PIC _ alpha*v_FLIP
    vec3 v_PIC(0,0,0), dv_FLIP(0,0,0);
    int rowSize = dim.z+1;
    int pageSize = (dim.y+1)*rowSize;
    for ( int i = minIndex.x; i <= maxIndex.x; ++i ) {
        vec3 d, s;
        d.x = i - gridIndex.x;
        d.x *= ( s.x = ( d.x < 0 ) ? -1.f : 1.f );
        int pageOffset = i*pageSize;
        for ( int j = minIndex.y; j <= maxIndex.y; ++j ) {
            d.y = j - gridIndex.y;
            d.y *= ( s.y = ( d.y < 0 ) ? -1.f : 1.f );
            int rowOffset = pageOffset + j*rowSize;
            for ( int k = minIndex.z; k <= maxIndex.z; ++k ) {
                d.z = k - gridIndex.z;
                d.z *= ( s.z = ( d.z < 0 ) ? -1.f : 1.f );
                const ParticleGrid::Node &node = nodes[rowOffset+k];
                float w;
                vec3 wg;
                weightAndGradient( s, d, w, wg );
                velocityGradient += mat3::outerProduct( node.velocity, wg );
                // Particle velocities
                v_PIC += node.velocity * w;
                dv_FLIP += node.velocityChange * w;
            }
        }
    }
    particle.velocity = (1.f-alpha)*v_PIC + alpha*(particle.velocity+dv_FLIP);
}

__device__ void updateParticleDeformationGradients( Particle &particle, const mat3 &velocityGradient, float timeStep, MaterialConstants *mat )
{
    // Temporarily assign all deformation to elastic portion
    mat3 F = mat3::addIdentity( timeStep*velocityGradient ) * particle.elasticF;

    // Clamp the singular values
    mat3 W, S, Sinv, V;
    computeSVD( F, W, S, V );

    S = mat3( CLAMP( S[0], mat->criticalCompression, mat->criticalStretch ), 0.f, 0.f,
              0.f, CLAMP( S[4], mat->criticalCompression, mat->criticalStretch ), 0.f,
              0.f, 0.f, CLAMP( S[8], mat->criticalCompression, mat->criticalStretch ) );
    Sinv = mat3( 1.f/S[0], 0.f, 0.f,
                 0.f, 1.f/S[4], 0.f,
                 0.f, 0.f, 1.f/S[8] );

    // Compute final deformation components
    particle.elasticF = mat3::multiplyADBt( W, S, V );
    particle.plasticF = mat3::multiplyADBt( V, Sinv, W ) * particle.plasticF;
}

// NOTE: assumes particleCount % blockDim.x = 0, so tid is never out of range!
// criticalCompression = 1 - theta_c
// criticalStretch = 1 + theta_s
__global__ void updateParticlesFromGrid( Particle *particles, Grid *grid, const ParticleGrid::Node *nodes, float timeStep, ImplicitCollider *colliders, int numColliders, MaterialConstants *mat )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    Particle &particle = particles[tid];

    // Update particle velocities and fill in velocity gradient for deformation gradient computation
    mat3 velocityGradient = mat3( 0.f );
    processGridVelocities( particle, grid, nodes, velocityGradient, 0.95f );

    updateParticleDeformationGradients( particle, velocityGradient, timeStep, mat );

    checkForAndHandleCollisions( colliders, numColliders, mat->coeffFriction, particle.position, particle.velocity );

    particle.position += timeStep * (particle.velocity );

}


void updateParticles( const SimulationParameters &parameters,
                      Particle *particles, int numParticles,
                      Grid *grid, ParticleGrid::Node *nodes, int numNodes, ParticleGridTempData *particleGridTempData,
                      ImplicitCollider *colliders, int numColliders,
                      MaterialConstants *mat )
{

    static const int threadCount = 128;

    computeParticleGridTempData<<< numParticles / threadCount , threadCount >>>( particles, grid, mat, particleGridTempData );
    checkCudaErrors( hipDeviceSynchronize() );

    // Clear grid data before update
    checkCudaErrors( hipMemset(nodes, 0, numNodes*sizeof(ParticleGrid::Node)) );

    dim3 blockDim = dim3(numParticles / threadCount, 64);
    dim3 threadDim = dim3(threadCount/64, 64);
    computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>( particles, grid, particleGridTempData, nodes );
    checkCudaErrors( hipDeviceSynchronize() );

    updateNodeVelocities<<< numNodes / threadCount, threadCount >>>( nodes, parameters.timeStep, colliders, numColliders, mat, grid, parameters.gravity );
    checkCudaErrors( hipDeviceSynchronize() );

    updateParticlesFromGrid<<< numParticles / threadCount, threadCount >>>( particles, grid, nodes, parameters.timeStep, colliders, numColliders, mat );
    checkCudaErrors( hipDeviceSynchronize() );

}
