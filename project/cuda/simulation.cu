#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   simulation.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 17 Apr 2014
**
**************************************************************************/

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "math.h"

#define CUDA_INCLUDE
#include "sim/caches.h"
#include "sim/collider.h"
#include "sim/material.h"
#include "sim/parameters.h"
#include "sim/particle.h"
#include "sim/particlegridnode.h"

#include "common/math.h"

#include "cuda/atomic.h"
#include "cuda/collider.h"
#include "cuda/decomposition.h"
#include "cuda/implicit.h"
#include "cuda/weighting.h"

#include "cuda/functions.h"

#define ALPHA 0.95f

// Chain to compute the volume of the particle

/**
 * Part of one time operation to compute particle volumes. First rasterize particle masses to grid
 *
 * Operation done over Particles over grid node particle affects
 */
__global__ void computeNodeMasses( const Particle *particles, const Grid *grid, float *nodeMasses )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    const Particle &particle = particles[particleIdx];

    glm::ivec3 currIJK;
    Grid::gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), currIJK );
    vec3 particleGridPos = (particle.position - grid->pos) / grid->h;
    currIJK += glm::ivec3(particleGridPos-1);

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        vec3 nodePosition(currIJK);
        vec3 dx = vec3::abs( particleGridPos - nodePosition );
        float w = weight( dx );
        atomicAdd( &nodeMasses[Grid::getGridIndex(currIJK, grid->dim+1)], particle.mass*w );
     }
}

/**
 * Computes the particle's density * grid's volume. This needs to be separate from computeCellMasses(...) because
 * we need to wait for ALL threads to sync before computing the density
 *
 * Operation done over Particles over grid node particle affects
 */
__global__ void computeParticleDensity( Particle *particles, const Grid *grid, const float *cellMasses )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    Particle &particle = particles[particleIdx];

    glm::ivec3 currIJK;
    Grid::gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), currIJK );
    vec3 particleGridPos = ( particle.position - grid->pos ) / grid->h;
    currIJK += glm::ivec3(particleGridPos-1);

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        vec3 nodePosition(currIJK);
        vec3 dx = vec3::abs( particleGridPos - nodePosition );
        float w = weight( dx );
        float gridVolume = grid->h * grid->h * grid->h;
        atomicAdd( &particle.volume, cellMasses[Grid::getGridIndex(currIJK, grid->dim+1)] * w / gridVolume ); //fill volume with particle density. Then in final step, compute volume
     }
}

/**
 * Computes the particle's volume. Assumes computeParticleDensity(...) has just been called.
 *
 * Operation done over particles
 */
__global__ void computeParticleVolume( Particle *particleData )
{
    int particleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    Particle &particle = particleData[particleIdx];
    particle.volume = particle.mass / particle.volume; // Note: particle.volume is assumed to be the (particle's density ) before we compute it correctly
}

__host__ void initializeParticleVolumes( Particle *particles, int numParticles, const Grid *grid, int numNodes )
{
    float *devNodeMasses;
    checkCudaErrors( hipMalloc( (void**)&devNodeMasses, numNodes*sizeof(float) ) );
    hipMemset( devNodeMasses, 0, numNodes*sizeof(float) );

    static const int threadCount = 128;

    dim3 blockDim = dim3( numParticles / threadCount, 64 );
    dim3 threadDim = dim3( threadCount/64, 64 );

    computeNodeMasses<<< blockDim, threadDim >>>( particles, grid, devNodeMasses );
    checkCudaErrors( hipDeviceSynchronize() );

    computeParticleDensity<<< blockDim, threadDim >>>( particles, grid, devNodeMasses );
    checkCudaErrors( hipDeviceSynchronize() );

    computeParticleVolume<<< numParticles / threadCount, threadCount >>>( particles );
    checkCudaErrors( hipDeviceSynchronize() );

    checkCudaErrors( hipFree(devNodeMasses) );
}

__global__ void computeSigma( const Particle *particles, ParticleCache *pCaches, const Grid *grid )
{
    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;

    const Particle &particle = particles[particleIdx];
    ParticleCache &pCache = pCaches[particleIdx];

    const mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    const mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re;
    computePD( Fe, Re );

    const MaterialConstants material = particle.material;

    float muFp = material.mu*__expf(material.xi*(1-Jpp));
    float lambdaFp = material.lambda*__expf(material.xi*(1-Jpp));

    pCache.sigma = (2*muFp*mat3::multiplyABt(Fe-Re, Fe) + mat3(lambdaFp*(Jep-1)*Jep)) * -particle.volume;
    pCache.particleGridPos = (particle.position - grid->pos)/grid->h;
}

/**
 * Called on each particle.
 *
 * Each particle adds it's mass, velocity and force contribution to the grid nodes within 2h of itself.
 *
 * In:
 * particleData -- list of particles
 * grid -- Stores grid paramters
 * worldParams -- Global parameters dealing with the physics of the world
 *
 * Out:
 * nodes -- list of every node in grid ((dim.x+1)*(dim.y+1)*(dim.z+1))
 *
 */
__global__ void computeCellMassVelocityAndForceFast( const Particle *particleData, const ParticleCache *pCaches, const Grid *grid, Node *nodes )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    const Particle &particle = particleData[particleIdx];
    const ParticleCache &pCache = pCaches[particleIdx];

    glm::ivec3 currIJK;
    Grid::gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    currIJK += glm::ivec3( pCache.particleGridPos-1 );

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        Node &node = nodes[Grid::getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient( pCache.particleGridPos - nodePosition, w, wg );

        atomicAdd( &node.mass, particle.mass*w );
        atomicAdd( &node.velocity, particle.velocity*particle.mass*w );
        atomicAdd( &node.force, pCache.sigma*wg );
     }
}

/**
 * Called on each grid node.
 *
 * Updates the velocities of each grid node based on forces and collisions
 *
 * In:
 * nodes -- list of all nodes in the grid.
 * dt -- delta time, time step of simulation
 * colliders -- array of colliders in the scene.
 * numColliders -- number of colliders in the scene
 * worldParams -- Global parameters dealing with the physics of the world
 * grid -- parameters defining the grid
 *
 * Out:
 * nodes -- updated velocity and velocityChange
 *
 */
__global__ void updateNodeVelocities( Node *nodes, float dt, const ImplicitCollider* colliders, int numColliders, const MaterialConstants *material, const Grid *grid )
{
    int nodeIdx = blockIdx.x*blockDim.x + threadIdx.x;
    Node &node = nodes[nodeIdx];

    if ( node.mass > 1e-12 ) {
        float scale = 1.f/node.mass;

        node.velocity *= scale; //Have to normalize velocity by mass to conserve momentum

        // Update velocity with node force
        vec3 tmpVelocity = node.velocity + dt*node.force*scale;

        // Handle collisions
        int gridI, gridJ, gridK;
        Grid::gridIndexToIJK(nodeIdx, gridI, gridJ, gridK, grid->dim+1);
        vec3 nodePosition = vec3(gridI, gridJ, gridK)*grid->h + grid->pos;
        checkForAndHandleCollisions( colliders, numColliders, material->coeffFriction, nodePosition, tmpVelocity );

        node.velocityChange = tmpVelocity - node.velocity;
        node.velocity = tmpVelocity;
    }
}

// Use weighting functions to compute particle velocity gradient and update particle velocity
__device__ void processGridVelocities( Particle &particle, const Grid *grid, const Node *nodes, mat3 &velocityGradient )
{
    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    // Compute neighborhood of particle in grid
    vec3 gridIndex = (pos - grid->pos) / h,
         gridMax = vec3::floor( gridIndex + vec3(2,2,2) ),
         gridMin = vec3::ceil( gridIndex - vec3(2,2,2) );
    glm::ivec3 maxIndex = glm::clamp( glm::ivec3(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( glm::ivec3(gridMin), glm::ivec3(0,0,0), dim );

    // For computing particle velocity gradient:
    //      grad(v_p) = sum( v_i * transpose(grad(w_ip)) ) = [3x3 matrix]
    // For updating particle velocity:
    //      v_PIC = sum( v_i * w_ip )
    //      v_FLIP = v_p + sum( dv_i * w_ip )
    //      v = (1-alpha)*v_PIC _ alpha*v_FLIP
    vec3 v_PIC(0,0,0), dv_FLIP(0,0,0);
    int rowSize = dim.z+1;
    int pageSize = (dim.y+1)*rowSize;
    for ( int i = minIndex.x; i <= maxIndex.x; ++i ) {
        vec3 d, s;
        d.x = gridIndex.x - i;
        d.x *= ( s.x = ( d.x < 0 ) ? -1.f : 1.f );
        int pageOffset = i*pageSize;
        for ( int j = minIndex.y; j <= maxIndex.y; ++j ) {
            d.y = gridIndex.y - j;
            d.y *= ( s.y = ( d.y < 0 ) ? -1.f : 1.f );
            int rowOffset = pageOffset + j*rowSize;
            for ( int k = minIndex.z; k <= maxIndex.z; ++k ) {
                d.z = gridIndex.z - k;
                d.z *= ( s.z = ( d.z < 0 ) ? -1.f : 1.f );
                const Node &node = nodes[rowOffset+k];
                float w;
                vec3 wg;
                weightAndGradient( -s, d, w, wg );
                velocityGradient += mat3::outerProduct( node.velocity, wg );
                // Particle velocities
                v_PIC += node.velocity * w;
                dv_FLIP += node.velocityChange * w;
            }
        }
    }
    particle.velocity = (1.f-ALPHA)*v_PIC + ALPHA*(particle.velocity+dv_FLIP);
}

__device__ void updateParticleDeformationGradients( Particle &particle, const mat3 &velocityGradient, float timeStep )
{
    // Temporarily assign all deformation to elastic portion
    particle.elasticF = mat3::addIdentity( timeStep*velocityGradient ) * particle.elasticF;
    const MaterialConstants mat = particle.material;
    // Clamp the singular values
    mat3 W, S, Sinv, V;
    computeSVD( particle.elasticF, W, S, V );

    // FAST COMPUTATION:

    S = mat3( CLAMP( S[0], mat.criticalCompression, mat.criticalStretch ), 0.f, 0.f,
              0.f, CLAMP( S[4], mat.criticalCompression, mat.criticalStretch ), 0.f,
              0.f, 0.f, CLAMP( S[8], mat.criticalCompression, mat.criticalStretch ) );

    Sinv = mat3( 1.f/S[0], 0.f, 0.f,
                 0.f, 1.f/S[4], 0.f,
                 0.f, 0.f, 1.f/S[8] );

    // Compute final deformation components
    particle.plasticF = mat3::multiplyADBt( V, Sinv, W ) * particle.elasticF * particle.plasticF;
    particle.elasticF = mat3::multiplyADBt( W, S, V );

//     // MORE ACCURATE COMPUTATION:

//    S[0] = CLAMP( S[0], mat->criticalCompression, mat->criticalStretch );
//    S[4] = CLAMP( S[4], mat->criticalCompression, mat->criticalStretch );
//    S[8] = CLAMP( S[8], mat->criticalCompression, mat->criticalStretch );

//    particle.elasticF = W * S * mat3::transpose( V );
//    particle.plasticF = V * mat3::inverse( S ) * mat3::transpose( W ) * particle.elasticF * particle.plasticF;

}

// NOTE: assumes particleCount % blockDim.x = 0, so tid is never out of range!
// criticalCompression = 1 - theta_c
// criticalStretch = 1 + theta_s
__global__ void updateParticlesFromGrid( Particle *particles, const Grid *grid, const Node *nodes, float timeStep, const ImplicitCollider *colliders, int numColliders, const MaterialConstants *mat, const vec3 gravity )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    Particle &particle = particles[tid];

    // Update particle velocities and fill in velocity gradient for deformation gradient computation
    mat3 velocityGradient = mat3( 0.f );
    processGridVelocities( particle, grid, nodes, velocityGradient );

    updateParticleDeformationGradients( particle, velocityGradient, timeStep );

    // Do this before collision test!
    particle.velocity += timeStep * gravity;

    checkForAndHandleCollisions( colliders, numColliders, mat->coeffFriction, particle.position, particle.velocity );

    particle.position += timeStep * ( particle.velocity );
}


/**
 * Approximate the shading normal of each particle
 * for each particle,find its corresponding grid node, then
 * approximate X,Y,Z component of mass gradient by examining average between the
 * two nearest neighbors on that axis (6 total - left,right,up,down,front,back)
 * then normalize the vector. If particle l2 norm is < epsilon (i.e. gradient is zero in all directions)
 * then just pick a random direction instead of normalizing.
 *
 * Particles that lie on boundary of surface will hopefully
 * have continuous normal.
 *
 * we could loop over grid nodes to cache the gradients at each node?
 * to reduce aliasing we could perturn the normals slightly. after all, snow is slightly scattery...
 */
__global__ void updateParticleNormals(Particle *particles, Grid *grid, const Node *nodes)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    Particle &particle = particles[tid];
    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    vec3 gridIndex = (pos - grid->pos) / h,
         gridMax = vec3::floor( gridIndex + vec3(1,1,1) ),
         gridMin = vec3::ceil( gridIndex - vec3(1,1,1) );
    glm::ivec3 maxIndex = glm::clamp( glm::ivec3(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( glm::ivec3(gridMin), glm::ivec3(0,0,0), dim );

    //glm::ivec3 ijk = glm::ivec3((pos - grid->pos) / h);


    // +x,-x,+y,-y,+z,-z axis-aligned components of negative gradient within grid
    // for higher resolution we could average over a larger neighborhood
    vec3 n = vec3(0,1,0);
//    ParticleGridNode &node;
//    int i1, i2; // grid indices of neighboring components
//    float c1,c2;
//    for (int a=0;a<3;a++)
//    {
//        glm::ivec3 da(0);
//        da[a]=-1;

//        da[1]=+1;
//        glm::ivec3 neighbor = ijk + glm::ivec3();

//        n[i] = (c1+c2)*.5;
//    }


//    //

    particle.normal = n;
}

__host__ void updateParticles( const SimulationParameters &parameters,
                               Particle *particles, ParticleCache *pCaches, int numParticles,
                               Grid *grid, Node *nodes, NodeCache *nodeCache, int numNodes,
                               ImplicitCollider *colliders, int numColliders,
                               MaterialConstants *material,
                               bool doShading)
{
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    static const int threadCount = 128;

    computeSigma<<< numParticles / threadCount , threadCount >>>( particles, pCaches, grid );
    checkCudaErrors( hipDeviceSynchronize() );

    // Clear grid data before update
    checkCudaErrors( hipMemset(nodes, 0, numNodes*sizeof(Node)) );

    dim3 blockDim = dim3( numParticles / threadCount, 64 );
    dim3 threadDim = dim3( threadCount/64, 64 );
    computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>( particles, pCaches, grid, nodes );
    checkCudaErrors( hipDeviceSynchronize() );

    if ( doShading )
    {
        updateParticleNormals<<< numParticles/threadCount, threadCount >>>( particles, grid, nodes );
        checkCudaErrors( hipDeviceSynchronize() );
    }

    updateNodeVelocities<<< numNodes / threadCount, threadCount >>>( nodes, parameters.timeStep, colliders, numColliders, material, grid );
    checkCudaErrors( hipDeviceSynchronize() );

    updateNodeVelocitiesImplicit( particles, pCaches, numParticles, grid, nodes, nodeCache, numNodes, parameters.timeStep, material );

    updateParticlesFromGrid<<< numParticles / threadCount, threadCount >>>( particles, grid, nodes, parameters.timeStep, colliders, numColliders, material, parameters.gravity );
    checkCudaErrors( hipDeviceSynchronize() );
}
