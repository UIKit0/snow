#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   simulation.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 17 Apr 2014
**
**************************************************************************/

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "math.h"

#define CUDA_INCLUDE
#include "sim/collider.h"
#include "sim/material.h"
#include "sim/parameters.h"
#include "sim/particle.h"
#include "sim/particlegridnode.h"

#include "common/math.h"

#include "cuda/collider.cu"
#include "cuda/decomposition.cu"
#include "cuda/weighting.cu"

#include "cuda/functions.h"

__host__ __device__ __forceinline__
bool withinBoundsInclusive( const float &v, const float &min, const float &max )
{
    return ( v >= min && v <= max );
}

__host__ __device__ __forceinline__
bool withinBoundsInclusive( const glm::ivec3 &v, const glm::ivec3 &min, const glm::ivec3 &max )
{
    return withinBoundsInclusive(v.x, min.x, max.x) && withinBoundsInclusive(v.y, min.y, max.y) && withinBoundsInclusive(v.z, min.z, max.z);
}


__host__ __device__ __forceinline__
void gridIndexToIJK( int idx, int &i, int &j, int &k,const  glm::ivec3 &nodeDim )
{
    i = idx / (nodeDim.y*nodeDim.z);
    idx = idx % (nodeDim.y*nodeDim.z);
    j = idx / nodeDim.z;
    k = idx % nodeDim.z;
}

__host__ __device__  __forceinline__
int getGridIndex( int i, int j, int k, const glm::ivec3 &nodeDim)
{
    return (i*(nodeDim.y*nodeDim.z) + j*(nodeDim.z) + k);
}

__host__ __device__ __forceinline__
void gridIndexToIJK( int idx, const  glm::ivec3 &nodeDim, glm::ivec3 &IJK )
{
    gridIndexToIJK(idx, IJK.x, IJK.y, IJK.z, nodeDim);
}

__host__ __device__ __forceinline__
int getGridIndex( const glm::ivec3 &IJK, const glm::ivec3 &nodeDim )
{
    return getGridIndex(IJK.x, IJK.y, IJK.z, nodeDim);
}


// Chain to compute the volume of the particle

/**
 * Part of one time operation to compute particle volumes. First rasterize particle masses to grid
 *
 * Operation done over Particles over grid node particle affects
 */
__global__ void computeCellMasses( Particle *particleData, Grid *grid, float* cellMasses )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particleData[particleIdx];

    glm::ivec3 currIJK;
    gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), currIJK );
    vec3 particleGridPos = (particle.position - grid->pos) / grid->h;
    currIJK.x += (int) particleGridPos.x - 1; currIJK.y += (int) particleGridPos.y - 1; currIJK.z += (int) particleGridPos.z - 1;

    if ( withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        vec3 nodePosition( currIJK.x, currIJK.y, currIJK.z );
        vec3 dx = vec3::abs( particleGridPos - nodePosition );
        float w = weight( dx );
        atomicAdd( &cellMasses[getGridIndex(currIJK, grid->dim+1)], particle.mass*w );
     }
}

/**
 * Computes the particle's density * grid's volume. This needs to be separate from computeCellMasses(...) because
 * we need to wait for ALL threads to sync before computing the density
 *
 * Operation done over Particles over grid node particle affects
 */
__global__ void computeParticleDensity( Particle *particleData, Grid *grid, float *cellMasses )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    Particle &particle = particleData[particleIdx];

    glm::ivec3 currIJK;
    gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), currIJK );
    vec3 particleGridPos = ( particle.position - grid->pos ) / grid->h;
    currIJK.x += (int) particleGridPos.x - 1; currIJK.y += (int) particleGridPos.y - 1; currIJK.z += (int) particleGridPos.z - 1;

    if ( withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ) {
        vec3 nodePosition( currIJK.x, currIJK.y, currIJK.z );
        vec3 dx = vec3::abs( particleGridPos - nodePosition );
        float w = weight( dx );
        float gridVolume = grid->h * grid->h * grid->h;
        atomicAdd( &particle.volume, cellMasses[getGridIndex(currIJK, grid->dim+1)] * w / gridVolume ); //fill volume with particle density. Then in final step, compute volume
     }
}

/**
 * Computes the particle's volume. Assumes computeParticleDensity(...) has just been called.
 *
 * Operation done over particles
 */
__global__ void computeParticleVolume( Particle *particleData )
{
    int particleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    Particle &particle = particleData[particleIdx];
    particle.volume = particle.mass / particle.volume; // Note: particle.volume is assumed to be the (particle's density ) before we compute it correctly
}

__host__ void initializeParticleVolumes( Particle *particles, int numParticles, Grid *grid, int numNodes )
{
    float *devCellMasses;
    checkCudaErrors( hipMalloc( (void**)&devCellMasses, numNodes*sizeof(float) ) );
    hipMemset( devCellMasses, 0, numNodes*sizeof(float) );

    static const int threadCount = 128;

    dim3 blockDim = dim3( numParticles / threadCount, 64 );
    dim3 threadDim = dim3( threadCount/64, 64 );

    computeCellMasses<<< blockDim, threadDim >>>( particles, grid, devCellMasses );
    checkCudaErrors( hipDeviceSynchronize() );

    computeParticleDensity<<< blockDim, threadDim >>>( particles, grid, devCellMasses );
    checkCudaErrors( hipDeviceSynchronize() );

    computeParticleVolume<<< numParticles / threadCount, threadCount >>>( particles );
    checkCudaErrors( hipDeviceSynchronize() );

    checkCudaErrors( hipFree( devCellMasses) );
}




__device__ void computeSigma( Particle &particle, MaterialConstants *material, mat3 &sigma )
{
    mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re;
    computePD(Fe, Re);

    float muFp = material->mu*__expf(material->xi*(1-Jpp));
    float lambdaFp = material->lambda*__expf(material->xi*(1-Jpp));

//    sigma = (2*muFp*(Fe-Re)*mat3::transpose(Fe)+lambdaFp*(Jep-1)*Jep*mat3(1.0f)) * (particle.volume);
    sigma = (2*muFp*mat3::multiplyABt(Fe-Re, Fe) + mat3(lambdaFp*(Jep-1)*Jep)) * -particle.volume;
}


__global__ void computeParticleGridTempData ( Particle *particleData, Grid *grid, MaterialConstants *material, ParticleTempData *particleGridTempData )
{
    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particleData[particleIdx];
    ParticleTempData &pgtd = particleGridTempData[particleIdx];

    pgtd.particleGridPos = (particle.position - grid->pos)/grid->h;
    computeSigma(particle, material, pgtd.sigma);
}

__device__ __forceinline__
void atomicAdd( vec3 *add, vec3 toAdd )
{
    atomicAdd(&(add->x), toAdd.x);
    atomicAdd(&(add->y), toAdd.y);
    atomicAdd(&(add->z), toAdd.z);
}

__device__ __forceinline__
void atomicAdd( mat3 *add, mat3 toAdd )
{
    atomicAdd(&(add->data[0]), toAdd[0]);
    atomicAdd(&(add->data[1]), toAdd[1]);
    atomicAdd(&(add->data[2]), toAdd[2]);
    atomicAdd(&(add->data[3]), toAdd[3]);
    atomicAdd(&(add->data[4]), toAdd[4]);
    atomicAdd(&(add->data[5]), toAdd[5]);
    atomicAdd(&(add->data[6]), toAdd[6]);
    atomicAdd(&(add->data[7]), toAdd[7]);
    atomicAdd(&(add->data[8]), toAdd[8]);
}

/**
 * Called on each particle.
 *
 * Each particle adds it's mass, velocity and force contribution to the grid nodes within 2h of itself.
 *
 * In:
 * particleData -- list of particles
 * grid -- Stores grid paramters
 * worldParams -- Global parameters dealing with the physics of the world
 *
 * Out:
 * nodes -- list of every node in grid ((dim.x+1)*(dim.y+1)*(dim.z+1))
 *
 */
__global__ void computeCellMassVelocityAndForceFast( Particle *particleData, Grid *grid, ParticleTempData *particleGridTempData, ParticleGridNode *nodes )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particleData[particleIdx];
    ParticleTempData &pgtd = particleGridTempData[particleIdx];

    glm::ivec3 currIJK;
    gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    currIJK.x += (int) pgtd.particleGridPos.x - 1; currIJK.y += (int) pgtd.particleGridPos.y - 1; currIJK.z += (int) pgtd.particleGridPos.z - 1;

    if (withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim)){
        ParticleGridNode &node = nodes[getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient(pgtd.particleGridPos-nodePosition, w, wg);

        atomicAdd(&node.mass, particle.mass*w);
        atomicAdd(&node.velocity, particle.velocity*particle.mass*w );
        atomicAdd(&node.force, pgtd.sigma*wg);
     }
}

/**
 * Called on each grid node.
 *
 * Updates the velocities of each grid node based on forces and collisions
 *
 * In:
 * nodes -- list of all nodes in the grid.
 * dt -- delta time, time step of simulation
 * colliders -- array of colliders in the scene.
 * numColliders -- number of colliders in the scene
 * worldParams -- Global parameters dealing with the physics of the world
 * grid -- parameters defining the grid
 *
 * Out:
 * nodes -- updated velocity and velocityChange
 *
 */
__global__ void updateNodeVelocities( ParticleGridNode *nodes, float dt, ImplicitCollider* colliders, int numColliders, MaterialConstants *material, Grid *grid )
{
    int nodeIdx = blockIdx.x*blockDim.x + threadIdx.x;
    ParticleGridNode &node = nodes[nodeIdx];

    if (node.mass > 1e-12){
        float scale = 1.f/node.mass;

        node.velocity *= scale; //Have to normalize velocity by mass to conserve momentum

        // Update velocity with node force
        vec3 tmpVelocity = node.velocity + dt*node.force*scale;

        // Handle collisions
        int gridI, gridJ, gridK;
        gridIndexToIJK(nodeIdx, gridI, gridJ, gridK, grid->dim+1);
        vec3 nodePosition = vec3(gridI, gridJ, gridK)*grid->h + grid->pos;
        checkForAndHandleCollisions( colliders, numColliders, material->coeffFriction, nodePosition, tmpVelocity );

        node.velocityChange = tmpVelocity - node.velocity;
        node.velocity = tmpVelocity;
    }
}

#define VEC2IVEC( V ) ( glm::ivec3((int)V.x, (int)V.y, (int)V.z) )

// Use weighting functions to compute particle velocity gradient and update particle velocity
__device__ void processGridVelocities( Particle &particle, Grid *grid, const ParticleGridNode *nodes, mat3 &velocityGradient, float alpha )
{
    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    // Compute neighborhood of particle in grid
    vec3 gridIndex = (pos - grid->pos) / h,
         gridMax = vec3::floor( gridIndex + vec3(2,2,2) ),
         gridMin = vec3::ceil( gridIndex - vec3(2,2,2) );
    glm::ivec3 maxIndex = glm::clamp( VEC2IVEC(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( VEC2IVEC(gridMin), glm::ivec3(0,0,0), dim );

    // For computing particle velocity gradient:
    //      grad(v_p) = sum( v_i * transpose(grad(w_ip)) ) = [3x3 matrix]
    // For updating particle velocity:
    //      v_PIC = sum( v_i * w_ip )
    //      v_FLIP = v_p + sum( dv_i * w_ip )
    //      v = (1-alpha)*v_PIC _ alpha*v_FLIP
    vec3 v_PIC(0,0,0), dv_FLIP(0,0,0);
    int rowSize = dim.z+1;
    int pageSize = (dim.y+1)*rowSize;
    for ( int i = minIndex.x; i <= maxIndex.x; ++i ) {
        vec3 d, s;
        d.x = gridIndex.x - i;
        d.x *= ( s.x = ( d.x < 0 ) ? -1.f : 1.f );
        int pageOffset = i*pageSize;
        for ( int j = minIndex.y; j <= maxIndex.y; ++j ) {
            d.y = gridIndex.y - j;
            d.y *= ( s.y = ( d.y < 0 ) ? -1.f : 1.f );
            int rowOffset = pageOffset + j*rowSize;
            for ( int k = minIndex.z; k <= maxIndex.z; ++k ) {
                d.z = gridIndex.z - k;
                d.z *= ( s.z = ( d.z < 0 ) ? -1.f : 1.f );
                const ParticleGridNode &node = nodes[rowOffset+k];
                float w;
                vec3 wg;
                weightAndGradient( -s, d, w, wg );
                velocityGradient += mat3::outerProduct( node.velocity, wg );
                // Particle velocities
                v_PIC += node.velocity * w;
                dv_FLIP += node.velocityChange * w;
            }
        }
    }
    particle.velocity = (1.f-alpha)*v_PIC + alpha*(particle.velocity+dv_FLIP);
}

__device__ void updateParticleDeformationGradients( Particle &particle, const mat3 &velocityGradient, float timeStep, MaterialConstants *mat )
{
    // Temporarily assign all deformation to elastic portion
    particle.elasticF = mat3::addIdentity( timeStep*velocityGradient ) * particle.elasticF;

    // Clamp the singular values
    mat3 W, S, Sinv, V;
    computeSVD( particle.elasticF, W, S, V );

    // FAST COMPUTATION:

    S = mat3( CLAMP( S[0], mat->criticalCompression, mat->criticalStretch ), 0.f, 0.f,
              0.f, CLAMP( S[4], mat->criticalCompression, mat->criticalStretch ), 0.f,
              0.f, 0.f, CLAMP( S[8], mat->criticalCompression, mat->criticalStretch ) );

    Sinv = mat3( 1.f/S[0], 0.f, 0.f,
                 0.f, 1.f/S[4], 0.f,
                 0.f, 0.f, 1.f/S[8] );

    // Compute final deformation components
    particle.plasticF = mat3::multiplyADBt( V, Sinv, W ) * particle.elasticF * particle.plasticF;
    particle.elasticF = mat3::multiplyADBt( W, S, V );

//     // MORE ACCURATE COMPUTATION:

//    S[0] = CLAMP( S[0], mat->criticalCompression, mat->criticalStretch );
//    S[4] = CLAMP( S[4], mat->criticalCompression, mat->criticalStretch );
//    S[8] = CLAMP( S[8], mat->criticalCompression, mat->criticalStretch );

//    particle.elasticF = W * S * mat3::transpose( V );
//    particle.plasticF = V * mat3::inverse( S ) * mat3::transpose( W ) * particle.elasticF * particle.plasticF;

}

// NOTE: assumes particleCount % blockDim.x = 0, so tid is never out of range!
// criticalCompression = 1 - theta_c
// criticalStretch = 1 + theta_s
__global__ void updateParticlesFromGrid( Particle *particles, Grid *grid, const ParticleGridNode *nodes, float timeStep, ImplicitCollider *colliders, int numColliders, MaterialConstants *mat, vec3 gravity )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    Particle &particle = particles[tid];

    // Update particle velocities and fill in velocity gradient for deformation gradient computation
    mat3 velocityGradient = mat3( 0.f );
    processGridVelocities( particle, grid, nodes, velocityGradient, 0.95f );

    updateParticleDeformationGradients( particle, velocityGradient, timeStep, mat );

    // Do this before collision test!
    particle.velocity += timeStep * gravity;

    checkForAndHandleCollisions( colliders, numColliders, mat->coeffFriction, particle.position, particle.velocity );

    particle.position += timeStep * ( particle.velocity );
}


/**
 * Approximate the shading normal of each particle
 * for each particle,find its corresponding grid node, then
 * approximate X,Y,Z component of mass gradient by examining average between the
 * two nearest neighbors on that axis (6 total - left,right,up,down,front,back)
 * then normalize the vector. If particle l2 norm is < epsilon (i.e. gradient is zero in all directions)
 * then just pick a random direction instead of normalizing.
 *
 * Particles that lie on boundary of surface will hopefully
 * have continuous normal.
 *
 * we could loop over grid nodes to cache the gradients at each node?
 * to reduce aliasing we could perturn the normals slightly. after all, snow is slightly scattery...
 */
__global__ void updateParticleNormals(Particle *particles, Grid *grid, const ParticleGridNode *nodes)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    Particle &particle = particles[tid];
    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    vec3 gridIndex = (pos - grid->pos) / h,
         gridMax = vec3::floor( gridIndex + vec3(1,1,1) ),
         gridMin = vec3::ceil( gridIndex - vec3(1,1,1) );
    glm::ivec3 maxIndex = glm::clamp( VEC2IVEC(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( VEC2IVEC(gridMin), glm::ivec3(0,0,0), dim );

    //glm::ivec3 ijk = VEC2IVEC((pos - grid->pos) / h);


    // +x,-x,+y,-y,+z,-z axis-aligned components of negative gradient within grid
    // for higher resolution we could average over a larger neighborhood
    vec3 n = vec3(0,1,0);
//    ParticleGridNode &node;
//    int i1, i2; // grid indices of neighboring components
//    float c1,c2;
//    for (int a=0;a<3;a++)
//    {
//        glm::ivec3 da(0);
//        da[a]=-1;

//        da[1]=+1;
//        glm::ivec3 neighbor = ijk + glm::ivec3();

//        n[i] = (c1+c2)*.5;
//    }


//    //


    particle.normal = n;
}

/**
 * Called over particles over nodes the particle affects. (numParticles * 64)
 *
 * Recommended:
 *  dim3 blockDim = dim3(numParticles / threadCount, 64);
 *  dim3 threadDim = dim3(threadCount/64, 64);
 *
 **/
__global__ void computedF(Particle *particles, Grid *grid, vec3 *du, mat3 *dFs){
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particles[particleIdx];
    mat3 &dF = dFs[particleIdx];

    vec3 particleGridPos = (particle.position - grid->pos)/grid->h;
    glm::ivec3 currIJK;
    gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    currIJK.x += (int) particleGridPos.x - 1; currIJK.y += (int) particleGridPos.y - 1; currIJK.z += (int) particleGridPos.z - 1;

    if (withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim)){
        vec3 du_j = du[getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient(particleGridPos-nodePosition, w, wg);

        atomicAdd(&dF, mat3::outerProduct(du_j, wg) * particle.elasticF);
     }

}

__device__ void computedR(mat3 &df, mat3 &Se, mat3 &Re, mat3 &dR){
    mat3 V = mat3::multiplyAtB(Re, dF) - mat3::multiplyAtB(dF, Re);

    // Solve for compontents of R^T * dR
    mat3 S = mat3(S[0]+S[4], S[5], -S[2], //remember, column major
                  S[5], S[0]+S[8], S[1],
                  -S[2], S[1], S[4]+S[8]);


    vec3 b(V[3], V[6], V[7]);

    vec3 x = mat3::inverse(S) * b; // Should replace this with a linear system solver function

    // Fill R^T * dR
    mat3 RTdR = mat3( 0, -x.x, -x.y, //remember, column major
                      x.x, 0, -x.z,
                      x.y, x.z, 0);

    dR = Re*RTdR;
}

// We will want to cache Re and Se since we will use it many times per time step
__global__ void computeAp(Particle *particles, Grid *grid, vec3 *du, mat3 *dFs, mat3 *Aps){
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particles[particleIdx];
    mat3 &dF = dFs[particleIdx];
    mat3 &Ap = Aps[particleIdx];

    mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re, Se;
    computePD(Fe, Re, Se);

    float muFp = material->mu*__expf(material->xi*(1-Jpp));
    float lambdaFp = material->lambda*__expf(material->xi*(1-Jpp));

    mat3 dRe = Re; // Need to actually compute dRe

    mat3 jFe_invTrans = Jep*mat3::transpose(mat3::inverse(Fe));


    Ap = (2*muFp*(dF - dRe) +lambdaFp*jFe_invTrans*mat3::innerProduct(jFe_invTrans, dF) + lambdaFp*(Jep - 1));

//    sigma = (2*muFp*(Fe-Re)*mat3::transpose(Fe)+lambdaFp*(Jep-1)*Jep*mat3(1.0f)) * (particle.volume);
//    sigma = (2*muFp*mat3::multiplyABt(Fe-Re, Fe) + mat3(lambdaFp*(Jep-1)*Jep)) * -particle.volume;
}

void updateParticles( const SimulationParameters &parameters,
                      Particle *particles, int numParticles,
                      Grid *grid, ParticleGridNode *nodes, int numNodes, ParticleTempData *particleGridTempData,
                      ImplicitCollider *colliders, int numColliders,
                      MaterialConstants *mat,
                      bool doShading)
{
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    static const int threadCount = 128;

    computeParticleGridTempData<<< numParticles / threadCount , threadCount >>>( particles, grid, mat, particleGridTempData );
    checkCudaErrors( hipDeviceSynchronize() );

    // Clear grid data before update
    checkCudaErrors( hipMemset(nodes, 0, numNodes*sizeof(ParticleGridNode)) );

    dim3 blockDim = dim3(numParticles / threadCount, 64);
    dim3 threadDim = dim3(threadCount/64, 64);
    computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>( particles, grid, particleGridTempData, nodes );
    checkCudaErrors( hipDeviceSynchronize() );

    if (doShading)
    {
        updateParticleNormals<<< numParticles/threadCount, threadCount >>> (particles, grid, nodes);
        checkCudaErrors( hipDeviceSynchronize() );
    }

    updateNodeVelocities<<< numNodes / threadCount, threadCount >>>( nodes, parameters.timeStep, colliders, numColliders, mat, grid );
    checkCudaErrors( hipDeviceSynchronize() );

    updateParticlesFromGrid<<< numParticles / threadCount, threadCount >>>( particles, grid, nodes, parameters.timeStep, colliders, numColliders, mat, parameters.gravity );
    checkCudaErrors( hipDeviceSynchronize() );
}
