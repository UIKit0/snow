#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   wil.cu
**   Author: mliberma
**   Created: 8 Apr 2014
**
**************************************************************************/

#ifndef WIL_CU
#define WIL_CU

#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> // prevents syntax errors on __global__ and __device__, among other things
#include "common/common.h"

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
#endif

extern "C"  {
    void testColliding();
    void testColliderNormal();
    void testGridMath();
    void timingTests();

}

// Grid math

#include "tim.cu" // should really be snow.cu or grid.cu depending on how we break it up
#include "decomposition.cu"
#include "weighting.cu"
#include "sim/particlegridnode.h"
#include "sim/material.h"
#include "cuda/collider.h"
#include "sim/particle.h"
//#include "matrix.cu"
//#include "vector.cu"

;


__host__ __device__ void computeSigma(Particle &particle, MaterialConstants *mat, mat3 &sigma){
    mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re;
    computePD(Fe, Re);

    float muFp = mat->mu*exp(mat->xi*(1-Jpp));
    float lambdaFp = mat->lambda*exp(mat->xi*(1-Jpp));

    sigma = (2*muFp*(Fe-Re)*mat3::transpose(Fe)+lambdaFp*(Jep-1)*Jep*mat3(1.0f)) * (particle.volume);
}


__device__ void atomicAdd(vec3 *add, vec3 toAdd){
    atomicAdd(&(add->x), toAdd.x);
    atomicAdd(&(add->y), toAdd.y);
    atomicAdd(&(add->z), toAdd.z);
}

/**
 * Called on each particle.
 *
 * Each particle adds it's mass, velocity and force contribution to the grid nodes within 2h of itself.
 *
 * In:
 * particleData -- list of particles
 * grid -- Stores grid paramters
 * MaterialConstants -- Global parameters dealing with the physics of the world
 *
 * Out:
 * nodes -- list of every node in grid ((dim.x+1)*(dim.y+1)*(dim.z+1))
 *
 */
__global__ void computeCellMassVelocityAndForce(Particle *particleData, Grid *grid, MaterialConstants *mat, ParticleGridNode *nodes){
    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;
    Particle &particle = particleData[particleIdx];

    mat3 sigma;
    computeSigma(particle, mat, sigma);

    vec3 particleGridPos = (particle.position - grid->pos)/grid->h;
    glm::ivec3 min = glm::ivec3(std::ceil(particleGridPos.x - 2), std::ceil(particleGridPos.y - 2), std::ceil(particleGridPos.z - 2));
    glm::ivec3 max = glm::ivec3(std::floor(particleGridPos.x + 2), std::floor(particleGridPos.y + 2), std::floor(particleGridPos.z + 2));


    // Apply particles contribution of mass, velocity and force to surrounding nodes
    min = glm::max(glm::ivec3(0.0f), min);
    max = glm::min(grid->dim, max);
    for (int i = min.x; i <= max.x; i++){
        for (int j = min.y; j <= max.y; j++){
            for (int k = min.z; k <= max.z; k++){
                int currIdx = getGridIndex(i, j, k, grid->dim+1);
                ParticleGridNode &node = nodes[currIdx];

                float w;
                vec3 wg;
                weightAndGradient(particleGridPos - vec3(i, j, k), w, wg);

                atomicAdd(&node.mass, particle.mass*w);
                atomicAdd(&node.velocity, particle.velocity*particle.mass*w);
                atomicAdd(&node.force, sigma*wg);
            }
        }
    }
}


__host__ __device__ __forceinline__
bool withinBoundsInclusive( const float &v, const float &min, const float &max ) { return (v >= min && v <= max); }

__host__ __device__ __forceinline__
bool withinBoundsInclusive( const glm::ivec3 &v, const glm::ivec3 &min, const glm::ivec3 &max ) { return  withinBoundsInclusive(v.x, min.x, max.x)
                                                                                                            && withinBoundsInclusive(v.y, min.y, max.y)
                                                                                                            && withinBoundsInclusive(v.z, min.z, max.z);}
__global__ void computeParticleGridTempData(Particle *particleData, Grid *grid, MaterialConstants *mat, ParticleTempData *particleGridTempData){
    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;
    Particle &particle = particleData[particleIdx];
    ParticleTempData &pgtd = particleGridTempData[particleIdx];


//    vec3 particleGridPos = (particle.position - grid->pos)/grid->h;
//    pgtd.gridIJK = glm::ivec3((int) particleGridPos.x, (int) particleGridPos.y, (int) particleGridPos.z);
    pgtd.particleGridPos = (particle.position - grid->pos)/grid->h;
    computeSigma(particle, mat, pgtd.sigma);
}

/**
 * Called on each particle.
 *
 * Each particle adds it's mass, velocity and force contribution to the grid nodes within 2h of itself.
 *
 * In:
 * particleData -- list of particles
 * grid -- Stores grid paramters
 * MaterialConstants -- Global parameters dealing with the physics of the world
 *
 * Out:
 * nodes -- list of every node in grid ((dim.x+1)*(dim.y+1)*(dim.z+1))
 *
 */
__global__ void computeCellMassVelocityAndForceFast(Particle *particleData, Grid *grid, ParticleTempData *particleGridTempData, ParticleGridNode *nodes){
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    Particle &particle = particleData[particleIdx];
    ParticleTempData &pgtd = particleGridTempData[particleIdx];

    glm::ivec3 currIJK;
    gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    currIJK.x += (int) pgtd.particleGridPos.x - 1; currIJK.y += (int) pgtd.particleGridPos.y - 1; currIJK.z += (int) pgtd.particleGridPos.z - 1;

    if (withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim)){
        ParticleGridNode &node = nodes[getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient(pgtd.particleGridPos-nodePosition, w, wg);

        atomicAdd(&node.mass, particle.mass*w);
        atomicAdd(&node.velocity, particle.velocity*particle.mass*w);
        atomicAdd(&node.force, pgtd.sigma*wg);
     }
}

/**
 * Called on each grid node.
 *
 * Updates the velocities of each grid node based on forces and collisions
 *
 * In:
 * nodes -- list of all nodes in the grid.
 * dt -- delta time, time step of simulation
 * colliders -- array of colliders in the scene.
 * numColliders -- number of colliders in the scene
 * MaterialConstants -- Global parameters dealing with the physics of the world
 * grid -- parameters defining the grid
 *
 * Out:
 * nodes -- updated velocity and velocityChange
 *
 */
__global__ void updateVelocities(ParticleGridNode *nodes, float dt, ImplicitCollider* colliders, int numColliders, MaterialConstants *mat, Grid *grid){
    int nodeIdx = blockIdx.x*blockDim.x + threadIdx.x;
    int gridI, gridJ, gridK;
    gridIndexToIJK(nodeIdx, gridI, gridJ, gridK, grid->dim+1);
    ParticleGridNode &node = nodes[nodeIdx];
    vec3 nodePosition = vec3(gridI, gridJ, gridK)*grid->h + grid->pos;

    node.velocity /= node.mass; //Have to normalize velocity by mass to conserve momentum

    vec3 tmpVelocity = node.velocity + dt*(node.force/node.mass);
    checkForAndHandleCollisions(colliders, numColliders, mat->coeffFriction, nodePosition, tmpVelocity);
    node.velocityChange = tmpVelocity - node.velocity;
    node.velocity = tmpVelocity;
}

/**
 * Updates the grid's nodes for this time step. First computes the mass, velocity and force acting on the grid
 * using a kernel over the particles and then updates the velocity in a second kernel over the grid nodes.
 * @param particleData
 * @param grid
 * @param MaterialConstants
 * @param nodes
 * @param dt
 * @param colliders
 * @param numColliders
 */
void gridMath(Particle *particleData, int numParticles, Grid *grid, MaterialConstants *mat, ParticleGridNode *nodes,
              float dt, ImplicitCollider* colliders, int numColliders, ParticleTempData *devPTGD){
    int threadCount = 256;
    computeParticleGridTempData<<< numParticles / threadCount , threadCount >>>(particleData, grid, mat, devPTGD);
    dim3 blockDim = dim3(numParticles / threadCount / 8, numParticles / threadCount / 8);
    dim3 threadDim = dim3(threadCount/64, 64);
    computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>(particleData, grid, devPTGD, nodes);
    updateVelocities<<< grid->nodeCount() / 256, 256 >>>(nodes, dt, colliders, numColliders, mat, grid);
}


// Begin testing code:

__global__ void testHalfPlaneColliding(){
    printf("\nTesting half plane colliding:\n");
    ImplicitCollider halfPlane;
    halfPlane.center = vec3(0,0,0);
    halfPlane.param = vec3(0,1,0);
    halfPlane.type = HALF_PLANE;

    if (isColliding(halfPlane, vec3(1,1,1))){ //expect no collision
        printf("\t[FAILED]: Simple non-colliding test on halfplane \n");
    } else{
        printf("\t[PASSED]: Simple non-colliding test on half plane \n");
    }
    if (!isColliding(halfPlane, vec3(-1,-1,-1))){ // expect collision
        printf("\t[FAILED]: Simple colliding test on halfplane failed\n");
    } else{
        printf("\t[PASSED]: Simple colliding test on half plane \n");
    }

    halfPlane.center = vec3(0,10,0);
    halfPlane.param = vec3(1,1,0);
    halfPlane.type = HALF_PLANE;

    if (isColliding(halfPlane, vec3(2,11,1))){ //expect no collision
        printf("\t[FAILED]: Non-colliding test on halfplane \n");
    } else{
        printf("\t[PASSED]: Non-colliding test on half plane \n");
    }
    if (!isColliding(halfPlane, vec3(-1,-1,-1))){ // expect collision
        printf("\t[FAILED]: Colliding test on halfplane \n");
    } else{
        printf("\t[PASSED]: Colliding test on half plane \n");
    }

    printf("Done testing half plane colliding\n\n");
}

__global__ void testSphereColliding(){
    printf("\nTesting sphere colliding:\n");
    ImplicitCollider sphereCollider;
    sphereCollider.center = vec3(0,0,0);
    sphereCollider.param = vec3(1,0,0);
    sphereCollider.type = SPHERE;

    if (isColliding(sphereCollider, vec3(1,1,1))){ //expect no collision
        printf("\t[FAILED]: Simple non-colliding test\n");
    } else{
        printf("\t[PASSED]: Simple non-colliding test\n");
    }
    if (!isColliding(sphereCollider, vec3(.5,0,0))){ // expect collision
        printf("\t[FAILED]: Simple colliding test\n");
    } else{
        printf("\t[PASSED]: Simple colliding test\n");
    }

    sphereCollider.center = vec3(0,10,0);
    sphereCollider.param = vec3(3.2,0,0);
    sphereCollider.type = SPHERE;

    if (isColliding(sphereCollider, vec3(0,0,0))){ //expect no collision
        printf("\t[FAILED]: Non-colliding test \n");
    } else{
        printf("\t[PASSED]: Non-colliding test \n");
    }
    if (!isColliding(sphereCollider, vec3(-1,10,-1))){ // expect collision
        printf("\t[FAILED]: Colliding test\n");
    } else{
        printf("\t[PASSED]: Colliding test\n");
    }


    printf("Done testing sphere colliding\n\n");
}

void testColliding(){
    testHalfPlaneColliding<<<1,1>>>();
    testSphereColliding<<<1,1>>>();
    hipDeviceSynchronize();
}

__host__ __device__ bool operator==(const vec3 &vecA, const vec3 &vecB)
{
   const double epsilion = 0.0001;  // choose something apprpriate.

   return    std::fabs(vecA[0] -vecB[0]) < epsilion
          && std::fabs(vecA[1] -vecB[1]) < epsilion
          && std::fabs(vecA[2] -vecB[2]) < epsilion;
}

__global__ void testHalfPlaneColliderNormal(){
    printf("\nTesting half plane colliderNormal:\n");
    ImplicitCollider halfPlane;
    halfPlane.center = vec3(0,0,0);
    halfPlane.param = vec3(0,1,0);
    halfPlane.type = HALF_PLANE;


    vec3 normal;
    vec3 expected = vec3(0,1,0);
    colliderNormal(halfPlane, vec3(1,-.001, 1), normal);
    if (normal == expected){
        printf("\t[PASSED]: Simple colliderNormal test on half plane \n");
    } else{
        printf("\t[FAILED]: Simple colliderNormal test on halfplane \n");
    }

    halfPlane.center = vec3(0,10,0);
    halfPlane.param = vec3(1,1,0);
    halfPlane.type = HALF_PLANE;

    expected = vec3(1,1,0);
    colliderNormal(halfPlane, vec3(0,9.999, 0), normal);
    if (expected == normal){
        printf("\t[PASSED]: colliderNormal test on half plane \n");
    } else{
        printf("\t[FAILED]: colliderNormal test on halfplane \n");
    }

    printf("Done testing half plane colliderNormal\n\n");
}

__global__ void testSphereColliderNormal(){
    printf("\nTesting sphere colliderNormal:\n");
    ImplicitCollider sphereCollider;
    sphereCollider.center = vec3(0,0,0);
    sphereCollider.param = vec3(1,0,0);
    sphereCollider.type = SPHERE;

    vec3 normal;
    vec3 expected = vec3::normalize(vec3(1.0f));
    colliderNormal(sphereCollider, vec3(.1f), normal);
    if (normal == expected){
        printf("\t[PASSED]: Simple colliderNormal test\n");
    } else{
        printf("\t[FAILED]: Simple colliderNormal test\n");
    }

    sphereCollider.center = vec3(0,10,0);
    sphereCollider.param = vec3(3.2,0,0);
    sphereCollider.type = SPHERE;

    expected = vec3(0,1,0);
    colliderNormal(sphereCollider, vec3(0,11,0), normal);
    if (normal == expected){
        printf("\t[PASSED]: colliderNormal test \n");
    } else{
       printf("\t[FAILED]: colliderNormal test \n");
    }

    printf("Done testing sphere colliderNormal\n\n");
}

void testColliderNormal(){
    testHalfPlaneColliderNormal<<<1,1>>>();
    testSphereColliderNormal<<<1,1>>>();
    hipDeviceSynchronize();
}

__device__ float maxNorm(mat3 m){
    float norm = -INFINITY;
    for (int row = 0; row < 3; row++){
        for (int col = 0; col < 3; col++){
            norm=max(norm,fabs(m[col*3+row])); //note glm is column major
        }
    }
    return norm;
}

__device__ void printMat3( const mat3 &mat ) {
    // prints by rows
    for (int j=0; j<3; ++j) // g3d stores column-major
    {
        for (int i=0; i<3; ++i)
        {
            printf("%f   ", mat[3*i+j]);
        }
        printf("\n");
    }
    printf("\n");
}


__global__ void testComputeSigma(){
    printf("\tTesting compute sigma \n");
    Particle p;
    p.mass = 1;
    p.elasticF = mat3(1.0f);
    p.plasticF = mat3(1.0f);
    p.volume = 1;

    MaterialConstants wp;
    wp.mu = 1;
    wp.lambda = 1;
    wp.xi = 1;

    mat3 sigma;
    computeSigma(p, &wp, sigma);

    mat3 expected = mat3(0.0f);

    if (maxNorm(expected-sigma) < 1e-4){
        printf("\t\t[PASSED]: Simple compute sigma\n");
    } else{
        printf("\t\t[FAILED]: Simple compute sigma\n");
    }

    // more complex test
    p.elasticF = mat3(1.f, -1.f, 3.f,
                      3.f, 0.f, 1.f,
                      2.f, 2.f, -1.f);
    p.plasticF = mat3(1.0f);

    computeSigma(p, &wp, sigma);

    expected = mat3( 130.7990,    4.5751,    6.5458,
                     4.5751,  116.4363,  -7.7044,
                     6.5458,   -7.7044,  125.9489);

    if (maxNorm(expected-sigma) < 1e-4){
        printf("\t\t[PASSED]: Complex compute sigma\n");
    } else{
        printf("\t\t[FAILED]: Complex compute sigma\n");
    }

    // even more complex test

    p.elasticF = mat3(0.6062,  0.3500, 0,
                           -0.3500, 0.6062, 0,
                           0,       0,      0.7000);
    p.plasticF = mat3( 0.9000,  0,  0,
                            0, 0.6364, 0.6364,
                            0,-0.6364, 0.6364);

    computeSigma(p, &wp, sigma);

    expected = mat3( -0.8462,    0.0000,         0,
                     0.0000,   -0.8462,        0,
                          0,         0,   -0.8462);

    if (maxNorm(expected-sigma) < 1e-4){
        printf("\t\t[PASSED]: More complex compute sigma\n");
    } else{
        printf("\t\t[FAILED]: More complex compute sigma\n");
    }

    printf("\tDone testing compute sigma \n");
}



__global__ void testCheckForAndHandleCollisions(){
    printf("\tTesting checkAndHandleCollisions\n");
    vec3 position = vec3(0,0,0);
    vec3 velocity = vec3(0,-1,-1);

    ImplicitCollider floor;
    floor.center = vec3(0,0,0);
    floor.param = vec3(0,1,0);
    floor.type = HALF_PLANE;

    ImplicitCollider colliders[1] = {floor};

    float coeffFriction = .5;

    checkForAndHandleCollisions(colliders, 1, coeffFriction, position, velocity);
    vec3 expected = vec3(0,0,-.5);

    if (velocity == expected){
        printf("\t\t[PASSED]: Simple checkAndHandleCollisions test\n");
    } else{
        printf("\t\t[FAILED]: Simple checkAndHandleCollisions test\n");
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    velocity = vec3(0,-1,-1);
    coeffFriction = 100000;
    checkForAndHandleCollisions(colliders, 1, coeffFriction, position, velocity);
    expected = vec3(0,0,0);

    if (velocity == expected){
        printf("\t\t[PASSED]: Simple high friction checkAndHandleCollisions test\n");
    } else{
        printf("\t\t[FAILED]: Simple high friction checkAndHandleCollisions test\n");
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    ImplicitCollider sphere;
    sphere.center = vec3(0,5,0);
    sphere.param = vec3(1,1,0);
    sphere.type = SPHERE;

    ImplicitCollider colliders2[2] = {floor, sphere};

    position = vec3(0,4,0);
    velocity = vec3(.5,1,-1);
    coeffFriction = .5;
    checkForAndHandleCollisions(colliders2, 2, coeffFriction, position, velocity);
    expected = vec3(.2764,0,-.5528);

    if (velocity == expected){
        printf("\t\t[PASSED]: Simple multiple colliders checkAndHandleCollisions test\n");
    } else{
        printf("\t\t[FAILED]: Simple multiple colliders checkAndHandleCollisions test\n");
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    position = vec3(0,4,0);
    velocity = vec3(.5,-1,-1);
    coeffFriction = .5;
    checkForAndHandleCollisions(colliders2, 2, coeffFriction, position, velocity);
    expected = vec3(.5,-1,-1);

    if (velocity == expected){
        printf("\t\t[PASSED]: Simple bodies are separating checkAndHandleCollisions test\n");
    } else{
        printf("\t\t[FAILED]: Simple bodies are separating checkAndHandleCollisions test\n");
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    position = vec3(0,100,0);
    velocity = vec3(.5,-1,-1);
    coeffFriction = .5;
    checkForAndHandleCollisions(colliders2, 2, coeffFriction, position, velocity);
    expected = vec3(.5,-1,-1);

    if (velocity == expected){
        printf("\t\t[PASSED]: Simple no collision checkAndHandleCollisions test\n");
    } else{
        printf("\t\t[FAILED]: Simple no collision checkAndHandleCollisions test\n");
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    printf("\tDone testing checkAndHandleCollisions\n");
    
}

#define NUM_PARTICLES 2

void testComputeCellMassVelocityAndForceComplex(){
    Particle particles[NUM_PARTICLES];
    for (int i = 0; i < NUM_PARTICLES; i++){
        particles[i].mass = i+1;
        particles[i].elasticF = mat3(1.0f);//Won't try more complicated values because we test sigma computation elsewhere
        particles[i].plasticF = mat3(1.0f);
        particles[i].velocity = vec3(i+1);
        particles[i].position = vec3(i);
        particles[i].volume = i+1;
    }

    Grid grid;
    grid.dim = glm::ivec3(5,5,5);
    grid.h = 1;
    grid.pos = vec3(-2,-2,-2);

    MaterialConstants wp;
    wp.lambda = wp.mu = wp.xi = 1;

    ParticleGridNode nodes[grid.nodeCount()];

    Particle *dev_particles;
    Grid *dev_grid;
    MaterialConstants *dev_wp;
    ParticleGridNode *dev_nodes;
    ParticleTempData *devPTGD;

    checkCudaErrors(hipMalloc((void**) &dev_particles, NUM_PARTICLES*sizeof(Particle)));
    checkCudaErrors(hipMemcpy(dev_particles,particles,NUM_PARTICLES*sizeof(Particle),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_grid, sizeof(Grid)));
    checkCudaErrors(hipMemcpy(dev_grid,&grid,sizeof(Grid),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_wp, sizeof(MaterialConstants)));
    checkCudaErrors(hipMemcpy(dev_wp,&wp,sizeof(MaterialConstants),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_nodes, grid.nodeCount()*sizeof(ParticleGridNode)));
    checkCudaErrors(hipMemcpy(dev_nodes,&nodes,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc( &devPTGD, NUM_PARTICLES*sizeof(ParticleTempData)));



//    computeCellMassVelocityAndForce<<<NUM_PARTICLES, 1>>>(dev_particles, dev_grid, dev_wp, dev_nodes);

    computeParticleGridTempData<<< NUM_PARTICLES, 1 >>>(dev_particles, dev_grid, dev_wp, devPTGD);
    dim3 blockDim = dim3(NUM_PARTICLES);
    dim3 threadDim = dim3(1, 64);
    computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>(dev_particles, dev_grid, devPTGD, dev_nodes);

    checkCudaErrors(hipMemcpy(nodes,dev_nodes,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyDeviceToHost));

    //I only check masses because the rest are derived from the same way mass is. The only one that is different is
    // force which I check the sigma function separately
    //These values are from the computeMasses.m file with this initial setup
    float expectedMasses[] ={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.00462962962962964,0.0185185185185185,0.00462962962962964,0,0,0,0.0185185185185185,0.0740740740740741,0.0185185185185185,0,0,0,0.00462962962962964,0.0185185185185185,0.00462962962962964,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.0185185185185185,0.0740740740740741,0.0185185185185185,0,0,0,0.0740740740740741,0.305555555555556,0.111111111111111,0.00925925925925927,0,0,0.0185185185185185,0.111111111111111,0.166666666666667,0.0370370370370371,0,0,0,0.00925925925925927,0.0370370370370371,0.00925925925925927,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.00462962962962964,0.0185185185185185,0.00462962962962964,0,0,0,0.0185185185185185,0.111111111111111,0.166666666666667,0.0370370370370371,0,0,0.00462962962962964,0.166666666666667,0.597222222222222,0.148148148148148,0,0,0,0.0370370370370371,0.148148148148148,0.0370370370370371,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.00925925925925927,0.0370370370370371,0.00925925925925927,0,0,0,0.0370370370370371,0.148148148148148,0.0370370370370371,0,0,0,0.00925925925925927,0.0370370370370371,0.00925925925925927,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0
};
    bool failed = false;
    for (int i =0; i < grid.nodeCount(); i++){
        int I,J,K;
        gridIndexToIJK(i, I, J, K, grid.dim+1);
        ParticleGridNode node = nodes[i];
        if ( std::abs(expectedMasses[i] - node.mass) > 1e-4){
            printf("\t\tActual mass (%f) didn't equal expected mass (%f) for node: (%d, %d, %d)\n", node.mass, expectedMasses[i], I,J,K);
            failed = true;
        }
        //printf("Node: ( %d, %d, %d), mass: %f\n", I,J,K, node.mass);
    }
    if (failed){
        printf("\t\t[FAILED]: Complex computeCellMassVelocityAndForce() test\n");
    }else{
        printf("\t\t[PASSED]: Complex computeCellMassVelocityAndForce() test\n");
    }


    hipFree(dev_particles);
    hipFree(dev_grid);
    hipFree(dev_wp);
    hipFree(dev_nodes);

    checkCudaErrors(hipDeviceSynchronize());
}

#define NUM_PARTICLES 2

void testComputeCellMassVelocityAndForce(){
    printf("\tTesting computeCellMassVelocityAndForce()\n");
    Particle particles[NUM_PARTICLES];
    for (int i = 0; i < NUM_PARTICLES; i++){
        particles[i].mass = i+1;
        particles[i].elasticF = mat3(1.0f);//Won't try more complicated values because we test sigma computation elsewhere
        particles[i].plasticF = mat3(1.0f);
        particles[i].velocity = vec3(i+1);
        particles[i].position = vec3(i);
        particles[i].volume = i+1;
    }

    Grid grid;
    grid.dim = glm::ivec3(1,1,1);
    grid.h = 1;
    grid.pos = vec3(0,0,0);

    MaterialConstants wp;
    wp.lambda = wp.mu = wp.xi = 1;

    ParticleGridNode nodes[grid.nodeCount()];

    Particle *dev_particles;
    Grid *dev_grid;
    MaterialConstants *dev_wp;
    ParticleGridNode *dev_nodes;
    ParticleTempData *devPTGD;

    checkCudaErrors(hipMalloc((void**) &dev_particles, NUM_PARTICLES*sizeof(Particle)));
    checkCudaErrors(hipMemcpy(dev_particles,particles,NUM_PARTICLES*sizeof(Particle),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_grid, sizeof(Grid)));
    checkCudaErrors(hipMemcpy(dev_grid,&grid,sizeof(Grid),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_wp, sizeof(MaterialConstants)));
    checkCudaErrors(hipMemcpy(dev_wp,&wp,sizeof(MaterialConstants),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_nodes, grid.nodeCount()*sizeof(ParticleGridNode)));
    checkCudaErrors(hipMemcpy(dev_nodes,&nodes,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc( &devPTGD, NUM_PARTICLES*sizeof(ParticleTempData)));



    //computeCellMassVelocityAndForce<<<NUM_PARTICLES, 1>>>(dev_particles, dev_grid, dev_wp, dev_nodes);

    computeParticleGridTempData<<< NUM_PARTICLES, 1 >>>(dev_particles, dev_grid, dev_wp, devPTGD);
    dim3 blockDim = dim3(NUM_PARTICLES);
    dim3 threadDim = dim3(1, 64);
    computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>(dev_particles, dev_grid, devPTGD, dev_nodes);

    checkCudaErrors(hipMemcpy(nodes,dev_nodes,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyDeviceToHost));

    //I only check masses because the rest are derived from the same way mass is. The only one that is different is
    // force which I check the sigma function separately
    //These values are from the computeMasses.m file with this initial setup
    float expectedMasses[8] ={.3056, .1111, .1111, .1667, .1111, .1667, .1667, .5972};
    bool failed = false;
    for (int i =0; i < grid.nodeCount(); i++){
        int I,J,K;
        gridIndexToIJK(i, I, J, K, grid.dim+1);
        ParticleGridNode node = nodes[i];
        if ( std::abs(expectedMasses[i] - node.mass) > 1e-4){
            printf("\t\tActual mass (%f) didn't equal expected mass (%f) for node: (%d, %d, %d)\n", node.mass, expectedMasses[i], I,J,K);
            failed = true;
        }
        //printf("Node: ( %d, %d, %d), mass: %f\n", I,J,K, node.mass);
    }
    if (failed){
        printf("\t\t[FAILED]: Simple computeCellMassVelocityAndForce() test\n");
    }else{
        printf("\t\t[PASSED]: Simple computeCellMassVelocityAndForce() test\n");
    }


    hipFree(dev_particles);
    hipFree(dev_grid);
    hipFree(dev_wp);
    hipFree(dev_nodes);

    checkCudaErrors(hipDeviceSynchronize());

    testComputeCellMassVelocityAndForceComplex();
    printf("\tDone testing computeCellMassVelocityAndForce()\n");
}


void testUpdateVelocities(){
    printf("\tTesting updateVelocities\n");

    Grid grid;
    grid.dim = glm::ivec3(1,0,0);
    grid.h = 1;

    float dt = 1;

    ParticleGridNode nodes2[2];
    //nodes[0].position = vec3(0,0,0), implicitly by index
    nodes2[0].mass = 1;
    nodes2[0].force = vec3(1,1,1);
    nodes2[0].velocity = vec3(0,1,0);

    //nodes[1].position = vec3(1,0,0), implicitly by index
    nodes2[1].mass = 2;
    nodes2[1].force = vec3(0,4,0);
    nodes2[1].velocity = vec3(0,0,0);


    ImplicitCollider sphere;
    sphere.center = vec3(1,1,0);
    sphere.param = vec3(1,0,0);
    sphere.type = SPHERE;
    ImplicitCollider colliders[1] = {sphere};

    MaterialConstants wp;
    wp.coeffFriction = .5;

    ImplicitCollider *dev_colliders;
    Grid *dev_grid;
    MaterialConstants *dev_wp;
    ParticleGridNode *dev_nodes;

    checkCudaErrors(hipMalloc((void**) &dev_colliders, sizeof(ImplicitCollider)));
    checkCudaErrors(hipMemcpy(dev_colliders,&colliders,sizeof(ImplicitCollider),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_grid, sizeof(Grid)));
    checkCudaErrors(hipMemcpy(dev_grid,&grid,sizeof(Grid),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_wp, sizeof(MaterialConstants)));
    checkCudaErrors(hipMemcpy(dev_wp,&wp,sizeof(MaterialConstants),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_nodes, grid.nodeCount()*sizeof(ParticleGridNode)));
    checkCudaErrors(hipMemcpy(dev_nodes,&nodes2,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyHostToDevice));

    updateVelocities<<<2,1>>>(dev_nodes, dt, dev_colliders, 1, dev_wp, dev_grid);

    checkCudaErrors(hipMemcpy(nodes2,dev_nodes,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyDeviceToHost));

    vec3 node0VExpected = vec3(1,2,1);
    if (nodes2[0].velocity == node0VExpected){
        printf("\t\t[PASSED]: Simple no collision updateVelocities test\n");
    } else{
        printf("\t\t[FAILED]: Simple no collision updateVelocities test\n");
        vec3 expected = node0VExpected;
        vec3 velocity = nodes2[0].velocity;
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    vec3 node1VExpected = vec3(0,0,0);
    if (nodes2[1].velocity == node1VExpected){
        printf("\t\t[PASSED]: Simple no collision updateVelocities test\n");
    } else{
        printf("\t\t[FAILED]: Simple no collision updateVelocities test\n");
        vec3 expected = node1VExpected;
        vec3 velocity = nodes2[1].velocity;
        printf("\t\t\tActual: (%f, %f, %f)  Expected: (%f, %f, %f)\n", velocity.x, velocity.y, velocity.z, expected.x, expected.y, expected.z);
    }

    hipFree(dev_colliders);
    hipFree(dev_grid);
    hipFree(dev_wp);
    hipFree(dev_nodes);

    printf("\tDone testing updateVelocities");
}

void testGridMath(){
    printf("\nTesting grid math\n");
    testComputeSigma<<<1,1>>>();
    hipDeviceSynchronize();

    testComputeCellMassVelocityAndForce();

    testCheckForAndHandleCollisions<<<1,1>>>();
    checkCudaErrors(hipDeviceSynchronize());

    testUpdateVelocities();


    printf("Done testing grid math\n");

}

void timingTests(){
    const int dim = 128;
    Grid grid;
    grid.dim = glm::ivec3( dim, dim, dim );
    grid.h = 1.f/dim;
    grid.pos = vec3(0,0,0);

    int nParticles = 5000*32;
    printf( "    Generating %d particles (%.2f MB)...\n",
            nParticles, nParticles*sizeof(Particle)/1e6 );
    fflush(stdout);
    Particle *particles = new Particle[nParticles];
    for ( int i = 0; i < nParticles; ++i ) {
        Particle particle;
        particle.position = grid.pos + vec3( urand(), urand(), urand() );
        particle.velocity = vec3( 0.f, -0.124f, 0.f );
        particle.elasticF = mat3(1.f);
        particle.plasticF = mat3(1.f);
        particles[i] = particle;
    }

    printf( "    Generating %d grid nodes (%.2f MB)...\n",
            (dim+1)*(dim+1)*(dim+1), (dim+1)*(dim+1)*(dim+1)*sizeof(ParticleGridNode)/1e6 );
    fflush(stdout);
    ParticleGridNode *nodes = new ParticleGridNode[grid.nodeCount()];
    for ( int i = 0; i <= dim; ++i ) {
        for ( int j = 0; j <= dim; ++j ) {
            for ( int k = 0; k <= dim; ++k ) {
                ParticleGridNode node;
                node.velocity = vec3( 0.f, 0.f, 0.f );
                node.velocityChange = vec3( 0.f, 0.f, 0.f );
                nodes[i*(dim+1)*(dim+1)+j*(dim+1)+k] = node;
            }
        }
    }

    MaterialConstants mat;
    mat.mu = 58333;
    mat.lambda = 38888;
    mat.xi = 10;
    mat.coeffFriction = .1;

    ImplicitCollider floor;
    floor.center = vec3(0,0,0);
    floor.param = vec3(0,1,0);
    floor.type = HALF_PLANE;

    ImplicitCollider colliders[] = {floor};
    int nColliders = 1;

    printf( "    Allocating kernel resources...\n" ); fflush(stdout);
    Particle *devParticles;
    ParticleGridNode *devNodes;
    Grid *devGrid;
    MaterialConstants *devMaterialConstants;
    ImplicitCollider *devColliders;
    ParticleTempData *devPTGD;
    checkCudaErrors(hipMalloc( &devParticles, nParticles*sizeof(Particle) ));
    checkCudaErrors(hipMalloc( &devNodes, (dim+1)*(dim+1)*(dim+1)*sizeof(ParticleGridNode) ));
    checkCudaErrors(hipMalloc( &devGrid, sizeof(Grid) ));
    checkCudaErrors(hipMalloc( &devMaterialConstants, sizeof(mat) ));
    checkCudaErrors(hipMalloc( &devColliders, nColliders*sizeof(ImplicitCollider) ));
    checkCudaErrors(hipMalloc( &devPTGD, nParticles*sizeof(ParticleTempData)));

    static const int blockSizes[] = { 64, 128, 256, 512 };
    static const int nBlocks = 4;

    float dt = .001;
    for ( int i = 0; i < nBlocks; ++i ) {
        checkCudaErrors(hipMemcpy( devParticles, particles, nParticles*sizeof(Particle), hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( devNodes, nodes, (dim+1)*(dim+1)*(dim+1)*sizeof(ParticleGridNode), hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( devGrid, &grid, sizeof(Grid), hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( devMaterialConstants, &mat, sizeof(mat), hipMemcpyHostToDevice ));
        checkCudaErrors(hipMemcpy( devColliders, colliders, nColliders*sizeof(ImplicitCollider), hipMemcpyHostToDevice ));
        int threadCount = blockSizes[i];
//        int blockSize = 256;
        printf( "    Block size = %3d; ", threadCount ); fflush(stdout);

        TIME( " Launching full kernel... ", "finished\n",

              computeParticleGridTempData<<< nParticles / threadCount , threadCount >>>(devParticles, devGrid, devMaterialConstants, devPTGD);
              dim3 blockDim = dim3(nParticles / threadCount, 64);
              dim3 threadDim = dim3(threadCount/64, 64);
              computeCellMassVelocityAndForceFast<<< blockDim, threadDim >>>(devParticles, devGrid, devPTGD, devNodes);
//              computeCellMassVelocityAndForce<<< nParticles / threadCount, threadCount >>>(devParticles, devGrid, devMaterialConstants, devNodes);
              updateVelocities<<< grid.nodeCount() / threadCount, threadCount >>>(devNodes, dt, devColliders, nColliders, devMaterialConstants, devGrid);
              checkCudaErrors(hipDeviceSynchronize());
        );
    }

//    checkCudaErrors(hipMemcpy(nodes,devNodes,grid.nodeCount()*sizeof(ParticleGridNode),hipMemcpyDeviceToHost));
//    printf("mass: %f", nodes[21489].mass);

    printf( "    Freeing kernel resources...\n" ); fflush(stdout);
    checkCudaErrors(hipFree( devParticles ));
    checkCudaErrors(hipFree( devNodes ));
    checkCudaErrors(hipFree( devGrid ));
    checkCudaErrors(hipFree( devMaterialConstants ));
    checkCudaErrors(hipFree( devColliders ));
    checkCudaErrors(hipFree( devPTGD ));
    delete [] particles;
    delete [] nodes;
}



// IT IS SLOW!
void testFillParticleVolume(){
    int numParticles = 128;
    Particle *particles = new Particle[numParticles];

    for (int i = 0; i < numParticles; i++){
        particles[i].mass = 1;
        particles[i].position = vec3(urand(), urand(), urand());
        particles[i].volume = 0;
    }

    Grid grid;
    int dim = 16;
    grid.dim = glm::ivec3(dim,dim,dim);
    grid.h = 1.0f/dim;

    float *cellMasses = new float[grid.nodeCount()];
    memset(cellMasses, 0, sizeof(float)*grid.nodeCount());

    printf("Computing cell masses...\n"); fflush(stdout);

    //Compute cell masses
    for (int nIdx = 0; nIdx < grid.nodeCount(); nIdx++){
        for (int pIdx = 0; pIdx < numParticles; pIdx++){
            Particle &particle = particles[pIdx];

            glm::ivec3 IJK;
            gridIndexToIJK(nIdx, grid.nodeDim(), IJK);


            vec3 nodePosition(IJK.x, IJK.y, IJK.z);
            vec3 particleGridPos = (particle.position - grid.pos)/grid.h;
            vec3 dx = vec3::abs(particleGridPos-nodePosition);
            float w = weight(dx);
            if (particle.mass > 1 || w > 10){
                printf("mass: %f, w: %f", particle.mass, w);
            }
            cellMasses[nIdx] += w*particle.mass;
        }
    }

    for (int i =0 ; i < grid.nodeCount(); i++){
        printf("cellMasses[i]: %f\n", cellMasses[i]);
    }

    printf("Computing particle volumes... \n"); fflush(stdout);

    //Compute volumes
    float *volumes = new float[numParticles];
    memset(volumes, 0, sizeof(float)*numParticles);

    for (int pIdx = 0; pIdx < numParticles; pIdx++){
        Particle &particle = particles[pIdx];
        for (int nIdx = 0; nIdx < grid.nodeCount(); nIdx++){
            glm::ivec3 IJK;
            gridIndexToIJK(nIdx, grid.nodeDim(), IJK);


            vec3 nodePosition(IJK.x, IJK.y, IJK.z);
            vec3 particleGridPos = (particle.position - grid.pos)/grid.h;
            vec3 dx = vec3::abs(particleGridPos-nodePosition);
            float w = weight(dx);
            volumes[pIdx] += cellMasses[nIdx]*w;
        }
        float gridVolume = grid.h*grid.h*grid.h;
        volumes[pIdx] = particle.mass / (volumes[pIdx] / gridVolume);
    }

     Particle *devParticles;
     Grid *devGrid;
     checkCudaErrors(hipMalloc( &devParticles, numParticles*sizeof(Particle) ));
     checkCudaErrors(hipMemcpy( devParticles, particles, numParticles*sizeof(Particle), hipMemcpyHostToDevice ));
     checkCudaErrors(hipMalloc( &devGrid, sizeof(Grid) ));
     checkCudaErrors(hipMemcpy( devGrid, &grid, sizeof(Grid), hipMemcpyHostToDevice ));

     printf("Calling fillParticleVolume kernel\n"); fflush(stdout);

     fillParticleVolume(devParticles, numParticles, devGrid, grid.nodeCount());

     printf("Comparing values\n"); fflush(stdout);

     checkCudaErrors(hipMemcpy( particles, devParticles, numParticles*sizeof(Particle), hipMemcpyDeviceToHost ));

     bool failed = false;
     for (int i = 0; i < numParticles; i++){
         if (std::fabs(volumes[i] - particles[i].volume) > 1e-8 || std::isnan(particles[i].volume)){
             failed = true;
//             printf("Expected: %f, Actual: %f\n", volumes[i], particles[i].volume); fflush(stdout);
         }
         printf("volume: %g, part.volume: %g\n", volumes[i], particles[i].volume);
     }

     if (failed){
         printf("[FAILED]: test fillParticleVolume Test\n");
     } else{
         printf("[PASSED]: test fillParticleVolume Test\n");
     }

     checkCudaErrors(hipFree( devParticles ));
     checkCudaErrors(hipFree( devGrid ));
     delete[] cellMasses;
     delete[] volumes;
}


#define NUM_PARTICLES 2
void testcomputeCellMasses(){
    Particle particles[NUM_PARTICLES];
    for (int i = 0; i < NUM_PARTICLES; i++){
        particles[i].mass = i+1;
        particles[i].elasticF = mat3(1.0f);//Won't try more complicated values because we test sigma computation elsewhere
        particles[i].plasticF = mat3(1.0f);
        particles[i].velocity = vec3(i+1);
        particles[i].position = vec3(i);
        particles[i].volume = 0;
    }

    Grid grid;
    grid.dim = glm::ivec3(5,5,5);
    grid.h = 1;
    grid.pos = vec3(-2,-2,-2);

    Particle *dev_particles;
    Grid *dev_grid;
    float *devCellMasses;

    checkCudaErrors(hipMalloc((void**) &dev_particles, NUM_PARTICLES*sizeof(Particle)));
    checkCudaErrors(hipMemcpy(dev_particles,particles,NUM_PARTICLES*sizeof(Particle),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**) &dev_grid, sizeof(Grid)));
    checkCudaErrors(hipMemcpy(dev_grid,&grid,sizeof(Grid),hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc( &devCellMasses, grid.nodeCount()*sizeof(float)));
    hipMemset(devCellMasses, 0, grid.nodeCount()*sizeof(float));


    dim3 blockDim = dim3(NUM_PARTICLES);
    dim3 threadDim = dim3(1, 64);
    computeCellMasses<<< blockDim, threadDim >>>(dev_particles, dev_grid, devCellMasses);

    float *cellMasses = new float[grid.nodeCount()];

    checkCudaErrors(hipMemcpy(cellMasses,devCellMasses,grid.nodeCount()*sizeof(float),hipMemcpyDeviceToHost));

    //I only check masses because the rest are derived from the same way mass is. The only one that is different is
    // force which I check the sigma function separately
    //These values are from the computeMasses.m file with this initial setup
    float expectedMasses[] ={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.00462962962962964,0.0185185185185185,0.00462962962962964,0,0,0,0.0185185185185185,0.0740740740740741,0.0185185185185185,0,0,0,0.00462962962962964,0.0185185185185185,0.00462962962962964,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.0185185185185185,0.0740740740740741,0.0185185185185185,0,0,0,0.0740740740740741,0.305555555555556,0.111111111111111,0.00925925925925927,0,0,0.0185185185185185,0.111111111111111,0.166666666666667,0.0370370370370371,0,0,0,0.00925925925925927,0.0370370370370371,0.00925925925925927,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.00462962962962964,0.0185185185185185,0.00462962962962964,0,0,0,0.0185185185185185,0.111111111111111,0.166666666666667,0.0370370370370371,0,0,0.00462962962962964,0.166666666666667,0.597222222222222,0.148148148148148,0,0,0,0.0370370370370371,0.148148148148148,0.0370370370370371,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0.00925925925925927,0.0370370370370371,0.00925925925925927,0,0,0,0.0370370370370371,0.148148148148148,0.0370370370370371,0,0,0,0.00925925925925927,0.0370370370370371,0.00925925925925927,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0
};
    bool failed = false;
    for (int i =0; i < grid.nodeCount(); i++){
        int I,J,K;
        gridIndexToIJK(i, I, J, K, grid.dim+1);
        if ( std::abs(expectedMasses[i] - cellMasses[i]) > 1e-4 || std::isnan(cellMasses[i])){
            printf("\t\tActual mass (%f) didn't equal expected mass (%f) for node: (%d, %d, %d), %d\n", cellMasses[i], expectedMasses[i], I,J,K, i); fflush(stdout);
            failed = true;
        }
//        printf("Node: ( %d, %d, %d), mass: %f, expected: %f\n", I,J,K, cellMasses[i], expectedMasses[i]);
    }
    if (failed){
        printf("\t\t[FAILED]: Complex computeCellMasses() test\n");
    }else{
        printf("\t\t[PASSED]: Complex computeCellMassse() test\n");
    }
checkCudaErrors(hipDeviceSynchronize());

    computeParticleDensity<<< blockDim, threadDim >>>(dev_particles, dev_grid, devCellMasses);

    checkCudaErrors(hipMemcpy(particles,dev_particles,NUM_PARTICLES*sizeof(Particle),hipMemcpyDeviceToHost));

    checkCudaErrors(hipDeviceSynchronize());

    for (int i = 0; i < NUM_PARTICLES; i++){
        printf("density: %f\n", particles[i].volume);
    }

    computeParticleVolume<<< NUM_PARTICLES, 1 >>>(dev_particles, dev_grid);

    checkCudaErrors(hipMemcpy(particles,dev_particles,NUM_PARTICLES*sizeof(Particle),hipMemcpyDeviceToHost));

    checkCudaErrors(hipDeviceSynchronize());

    for (int i = 0; i < NUM_PARTICLES; i++){
        printf("volumes: %f\n", particles[i].volume);
    }

    hipFree(dev_particles);
    hipFree(dev_grid);
    hipFree(devCellMasses);

    delete[] cellMasses;

    checkCudaErrors(hipDeviceSynchronize());

//    printf("\tDone testing computeCellMassVelocityAndForce()\n");

}



#endif // WIL_CU
