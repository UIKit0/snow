#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   wil.cu
**   Author: mliberma
**   Created: 8 Apr 2014
**
**************************************************************************/

#ifndef WIL_CU
#define WIL_CU

#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> // prevents syntax errors on __global__ and __device__, among other things
#include <glm/geometric.hpp>
#include "math.h"   // this imports the CUDA math library
#include "sim/collider.h"

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
#endif

extern "C"  {
    void testColliding();
}

typedef bool (*isCollidingFunc) (ImplicitCollider collider, glm::vec3 position);


// Apparent glm::dot doesn't work...
__device__ float dot(glm::vec3 v, glm::vec3 w){
    return v.x * w.x + v.y*w.y + v.z*w.z;
}

/**
 * A collision occurs when the point is on the OTHER side of the normal
 */
__device__ bool isCollidingHalfPlane(glm::vec3 planePoint, glm::vec3 planeNormal, glm::vec3 position){
    glm::vec3 vecToPoint = position - planePoint;
    return (dot(vecToPoint, planeNormal) < 0);
}

/**
 * Defines a halfplane such that collider.center is a point on the plane,
 * and collider.param is the normal to the plane.
 */
__device__ bool isCollidingHalfPlaneImplicit(ImplicitCollider collider, glm::vec3 position){
    return isCollidingHalfPlane(collider.center, collider.param, position);
}

/** array of colliding functions. isCollidingFunctions[collider.type] will be the correct function */
__device__ isCollidingFunc isCollidingFunctions[1] = {isCollidingHalfPlaneImplicit};

/**
 * General purpose function for handling colliders
 */
__device__ bool isColliding(ImplicitCollider collider, glm::vec3 position){
    return isCollidingFunctions[collider.type](collider, position);
}

// Begin testing code:

__global__ void testHalfPlaneColliding(){
    printf("\nTesting half plane colliding:\n");
    ImplicitCollider halfPlane;
    halfPlane.center = glm::vec3(0,0,0);
    halfPlane.param = glm::vec3(0,1,0);
    halfPlane.type = HALF_PLANE;

    if (isColliding(halfPlane, glm::vec3(1,1,1))){ //expect no collision
        printf("\t[FAILED]: Simple non-colliding test on halfplane \n");
    } else{
        printf("\t[PASSED]: Simple non-colliding test on half plane \n");
    }
    if (!isColliding(halfPlane, glm::vec3(-1,-1,-1))){ // expect collision
        printf("\t[FAILED]: Simple colliding test on halfplane failed\n");
    } else{
        printf("\t[PASSED]: Simple colliding test on half plane \n");
    }

    halfPlane.center = glm::vec3(0,10,0);
    halfPlane.param = glm::vec3(1,1,0);
    halfPlane.type = HALF_PLANE;

    if (isColliding(halfPlane, glm::vec3(2,11,1))){ //expect no collision
        printf("\t[FAILED]: Non-colliding test on halfplane \n");
    } else{
        printf("\t[PASSED]: Non-colliding test on half plane \n");
    }
    if (!isColliding(halfPlane, glm::vec3(-1,-1,-1))){ // expect collision
        printf("\t[FAILED]: Colliding test on halfplane failed\n");
    } else{
        printf("\t[PASSED]: Colliding test on half plane \n");
    }


    printf("Done testing half plane colliding\n\n");
}

void testColliding(){
    testHalfPlaneColliding<<<1,1>>>();
    hipDeviceSynchronize();
}

#endif // WIL_CU
