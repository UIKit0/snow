#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   decomposition.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 13 Apr 2014
**
**************************************************************************/

#ifndef DECOMPOSITION_H
#define DECOMPOSITION_H

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common/math.h"

#include "cuda/matrix.cu"
#include "cuda/vector.cu"
#include "cuda/quaternion.cu"

#define GAMMA 5.828427124 // FOUR_GAMMA_SQUARED = sqrt(8)+3;
#define CSTAR 0.923879532 // cos(pi/8)
#define SSTAR 0.3826834323 // sin(p/8)

__host__ __device__ void jacobiConjugation( int p, int q, mat3 &S, quat &qV )
{
    // eliminate off-diagonal entries Spq, Sqp
    float ch = 2.f * (S[0]-S[4]), ch2 = ch*ch;
    float sh = S[3], sh2 = sh*sh;
    bool flag = ( GAMMA * sh2 < ch2 );
    float w = rsqrt( ch2 + sh2 );
    ch = flag ? w*ch : CSTAR; ch2 = ch*ch;
    sh = flag ? w*sh : SSTAR; sh2 = sh*sh;

    // build rotation matrix Q
    float scale = ch*ch + sh*sh;
    float a = (ch2-sh2) / scale;
    float b = (2.f*sh*ch) / scale;
    mat3 Q( a, b, 0, -b, a, 0, 0, 0, 1 );

    // perform the conjugation to annihilate S = Q' S Q
    S = mat3::multiplyTransposeL( Q, S ) * Q;
    vec3 tmp( qV.x, qV.y, qV.z );
    tmp *= sh;
    sh *= qV.w;
    // original
    qV *= ch;

    // terrible hack, this arranges such that for
    // (p,q) = ((0,1),(1,2),(0,2)), n = (0,1,2)
    int n = 2*q-p-2;
    int x = n;
//    int y = (n+1) % 3;
//    int z = (n+2) % 3;
    int y = ( n == 2 ) ? 0 : n+1;
    int z = ( n > 0 ) ? n-1 : 2;
    qV[z] += sh;
    qV.w -= tmp[z];
    qV[x] += tmp[y];
    qV[y] -= tmp[x];

    // re-arrange matrix for next iteration
    S = mat3( S[4], S[5], S[3],
              S[5], S[8], S[2],
              S[3], S[2], S[0] );
}

/*
 * Wrapper function for the first step. Solve symmetric
 * eigenproblem using jacobi iteration. Given a symmetric
 * matrix S, diagonalize it also returns the cumulative
 * rotation as a quaternion.
 */
__host__ __device__ void jacobiEigenanalysis( mat3 &S, quat &qV )
{
    qV = glm::quat(1,0,0,0);
    for ( int sweep = 0; sweep < 4; ++sweep ) {
        // we wish to eliminate the maximum off-diagonal element
        // on every iteration, but cycling over all 3 possible rotations
        // in fixed order (p,q) = (1,2) , (2,3), (1,3) still has
        // asymptotic convergence
        jacobiConjugation( 0, 1, S, qV );
        jacobiConjugation( 1, 2, S, qV );
        jacobiConjugation( 0, 2, S, qV );
    }
}

// glm::toMat3 doesn't work in CUDA
__host__ __device__ __forceinline__ void toMat3( const quat &q, mat3 &M )
{
    float qxx = q.x*q.x;
    float qyy = q.y*q.y;
    float qzz = q.z*q.z;
    float qxz = q.x*q.z;
    float qxy = q.x*q.y;
    float qyz = q.y*q.z;
    float qwx = q.w*q.x;
    float qwy = q.w*q.y;
    float qwz = q.w*q.z;
    M[0] = 1.f - 2.f*(qyy+qzz);
    M[1] = 2.f * (qxy+qwz);
    M[2] = 2.f * (qxz-qwy);
    M[3] = 2.f * (qxy-qwz);
    M[4] = 1.f - 2.f*(qxx+qzz);
    M[5] = 2.f * (qyz+qwx);
    M[6] = 2.f * (qxz+qwy);
    M[7] = 2.f * (qyz-qwx);
    M[8] = 1.f - 2.f*(qxx+qyy);
}

#define condSwap( COND, X, Y )          \
{                                       \
    __typeof__ (X) _X_ = X;             \
    X = COND ? Y : X;                   \
    Y = COND ? _X_ : Y;                 \
}

#define condNegSwap( COND, X, Y )       \
{                                       \
    __typeof__ (X) _X_ = -X;            \
    X = COND ? Y : X;                   \
    Y = COND ? _X_ : Y;                 \
}

__host__ __device__ void sortSingularValues( mat3 &B, mat3 &V )
{
    // used in step 2
    vec3 b1 = B.column(0); vec3 v1 = V.column(0);
    vec3 b2 = B.column(1); vec3 v2 = V.column(1);
    vec3 b3 = B.column(2); vec3 v3 = V.column(2);
    float rho1 = vec3::dot(b1,b1);
    float rho2 = vec3::dot(b2,b2);
    float rho3 = vec3::dot(b3,b3);
    bool c;

    c = rho1 < rho2;
    condNegSwap( c, b1, b2 ); 
    condNegSwap( c, v1, v2 );
    condSwap( c, rho1, rho2 );

    c = rho1 < rho3;
    condNegSwap( c, b1, b3 ); 
    condNegSwap( c, v1, v3 );
    condSwap( c, rho1, rho3 );

    c = rho2 < rho3;
    condNegSwap( c, b2, b3 ); 
    condNegSwap( c, v2, v3 );

    // re-build B,V
    B = mat3( b1, b2, b3 );
    V = mat3( v1, v2, v3 );
}

//inline float accurateRSQRT(float x)
//{
//    // TODO - something is wrong with this.
//    // used in step 3
//    /* Lomont 2003 */
//    //float y = glm::fastSqrt(x);
//    //return y * (3-x*y*y)/2;
//}

//inline float accurateSQRT(float x)
//{
//    return x * accurateRSQRT(x);
//}

__host__ __device__ void QRGivensQuaternion( float a1, float a2, float &ch, float &sh )
{
    /// TODO - if SVD isnt accurate enough, work on fixing accurateSQRT function here

    // a1 = pivot point on diagonal
    // a2 = lower triangular entry we want to annihilate

    // the authors be trippin, accurateSQRT doesn't work...
    //float rho = accurateSQRT(a1*a1 + a2*a2);
    float tmp = a1*a1 + a2*a2;
    float rho = tmp * rsqrt(tmp); // = sqrt(tmp)

    sh = rho > EPSILON ? a2 : 0;
    ch = fabsf(a1) + fmaxf( rho, EPSILON );
    bool b = a1 < 0;
    condSwap( b, sh, ch );
    //float w = glm::inversesqrt(ch*ch+sh*sh);
    //float w = glm::fastInverseSqrt(ch*ch+sh*sh);
    float w = rsqrt( ch*ch + sh*sh );

    ch *= w;
    sh *= w;
}

__host__ __device__ void QRDecomposition( const mat3 &B, mat3 &Q, mat3 &R )
{
    R = B;

    // QR decomposition of 3x3 matrices using Givens rotations to
    // eliminate elements B21, B31, B32
    quat qQ; // cumulative rotation
    quat qU; // each Givens rotation in quaternion form

    mat3 U;
    float ch, sh;

    // first givens rotation
    QRGivensQuaternion( R[0], R[1], ch, sh );
    qU = quat( ch, 0, 0, sh );
    U = mat3::fromQuat(qU);
//    toMat3( qU, U );
    R = mat3::multiplyTransposeL( U, R );

    // update cumulative rotation
    qQ *= qU;

    // second givens rotation
    QRGivensQuaternion( R[0], R[2], ch, sh );
    qU = quat( ch, 0, -sh, 0 );
    U = mat3::fromQuat(qU);
//    toMat3( qU, U );
    R = mat3::multiplyTransposeL( U, R );

    // update cumulative rotation
    qQ *= qU;

    // third Givens rotation
    QRGivensQuaternion( R[4], R[5], ch, sh );
    qU = quat( ch, sh, 0, 0 );
    U = mat3::fromQuat(qU);
//    toMat3( qU, U );
    R = mat3::multiplyTransposeL( U, R );

    // update cumulative rotation
    qQ *= qU;

    // qQ now contains final rotation for Q
    Q = mat3::fromQuat(qQ);
//    toMat3( qQ, Q );
}

/*
 * McAdams, Selle, Tamstorf, Teran, and Sifakis. Computing the Singular Value Decomposition of 3 x 3
 * matrices with minimal branching and elementary floating point operations
 * Computes SVD of 3x3 matrix A = W * S * V'
 */
__host__ __device__ void computeSVD( const mat3 &A, mat3 &W, mat3 &S, mat3 &V )
{
    // normal equations matrix
    mat3 ATA = mat3::multiplyTransposeL( A, A );

/// 2. Symmetric Eigenanlysis
    quat qV;
    jacobiEigenanalysis( ATA, qV );
    V = mat3::fromQuat(qV);
//    toMat3( qV, V );
    mat3 B = A * V;

/// 3. Sorting the singular values (find V)
    sortSingularValues( B, V );

/// 4. QR decomposition
    QRDecomposition( B, W, S );
}

/*
 * Returns polar decomposition of 3x3 matrix M where
 * M = Fe = Re * Se = U * P
 * U is an orthonormal matrix
 * S is symmetric positive semidefinite
 * Can get Polar Decomposition from SVD, see first section of http://en.wikipedia.org/wiki/Polar_decomposition
 */
__host__ __device__ void computePD( const mat3 &A, mat3 &U, mat3 &P )
{
    // U is unitary matrix (i.e. orthogonal/orthonormal)
    // P is positive semidefinite Hermitian matrix
    mat3 W, S, V;
    computeSVD( A, W, S, V );
    mat3 Vt = mat3::transpose(V);
    U = W * Vt;
    P = V * S * Vt;
}

/*
 * In snow we desire both SVD and polar decompositions simultaneously without
 * re-computing USV for polar.
 * here is a function that returns all the relevant values
 * SVD : A = W * S * V'
 * PD : A = U * P
 */
__host__ __device__ void computeSVDandPD( const mat3 &A, mat3 &W, mat3 &S, mat3 &V, mat3 &U, mat3 &P )
{
    computeSVD( A, W, S, V );
    mat3 Vt = mat3::transpose(V);
    U = W * Vt;
    P = V * S * Vt;
}

#endif // DECOMPOSITION_H
