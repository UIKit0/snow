#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   implicit.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 26 Apr 2014
**
**************************************************************************/

#ifndef IMPLICIT_H
#define IMPLICIT_H

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define CUDA_INCLUDE
#include "geometry/grid.h"
#include "sim/particle.h"
#include "sim/particlegridnode.h"
#include "cuda/vector.cu"

#include "cuda/atomic.cu"
#include "cuda/weighting.cu"

#define BETA 0.5

/**
 * Called over particles over nodes the particle affects. (numParticles * 64)
 *
 * Recommended:
 *  dim3 blockDim = dim3(numParticles / threadCount, 64);
 *  dim3 threadDim = dim3(threadCount/64, 64);
 *
 **/
__global__ void computedF( Particle *particles, Grid *grid, float dt, vec3 *u, mat3 *dFs )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particles[particleIdx];
    mat3 &dF = dFs[particleIdx];

    vec3 particleGridPos = (particle.position - grid->pos)/grid->h;
    glm::ivec3 currIJK;
    Grid::gridIndexToIJK(threadIdx.y, glm::ivec3(4,4,4), currIJK);
    currIJK.x += (int) particleGridPos.x - 1; currIJK.y += (int) particleGridPos.y - 1; currIJK.z += (int) particleGridPos.z - 1;

    if ( Grid::withinBoundsInclusive(currIJK, glm::ivec3(0,0,0), grid->dim) ){
        vec3 du_j = dt * u[Grid::getGridIndex(currIJK, grid->dim+1)];

        float w;
        vec3 wg;
        vec3 nodePosition(currIJK.x, currIJK.y, currIJK.z);
        weightAndGradient(particleGridPos-nodePosition, w, wg);

        atomicAdd(&dF, mat3::outerProduct(du_j, wg) * particle.elasticF);
     }

}

__global__ void computeAp( Particle *particles, Grid *grid, mat3 *dFs, mat3 *Aps )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particles[particleIdx];
    mat3 &dF = dFs[particleIdx];
    mat3 &Ap = Aps[particleIdx];

    mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    mat3 &Fe = particle.elasticF;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    mat3 Re;
    computePD(Fe, Re);

//    float muFp = material->mu*__expf(material->xi*(1-Jpp));
//    float lambdaFp = material->lambda*__expf(material->xi*(1-Jpp));

//    mat3 dRe = Re; // Need to actually compute dRe

//    mat3 jFe_invTrans = Jep*mat3::transpose(mat3::inverse(Fe));

//    Ap = (2*muFp*(dF - dRe) +lambdaFp*jFe_invTrans*mat3::innerProduct(jFe_invTrans, dF) + lambdaFp*(Jep - 1));

////    sigma = (2*muFp*(Fe-Re)*mat3::transpose(Fe)+lambdaFp*(Jep-1)*Jep*mat3(1.0f)) * (particle.volume);
////    sigma = (2*muFp*mat3::multiplyABt(Fe-Re, Fe) + mat3(lambdaFp*(Jep-1)*Jep)) * -particle.volume;
}

__global__ void computedf( Particle *particles, Grid *grid, mat3 *Aps, vec3 *dfs )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    const Particle &particle = particles[particleIdx];

    vec3 gridPos = (particle.position-grid->pos)/grid->h;
    glm::ivec3 ijk;
    Grid::gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), ijk );
    ijk += glm::ivec3( gridPos.x-1, gridPos.y-1, gridPos.z-1 );

    if ( Grid::withinBoundsInclusive(ijk, glm::ivec3(0,0,0), grid->dim) ) {

        vec3 wg;
        vec3 nodePos(ijk);
        weightGradient( gridPos-nodePos, wg );

        const mat3 &Ap = Aps[particleIdx];
        vec3 df = -particle.volume * mat3::multiplyABt( Ap, particle.elasticF ) * wg;

        atomicAdd( &dfs[Grid::getGridIndex(ijk,grid->nodeDim())], df );
    }
}

__global__ void computeResult( ParticleGridNode *nodes, int numNodes, float dt, const vec3 *u, const vec3 *dfs, vec3 *result )
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if ( tid >= numNodes ) return;
    result[tid] = u[tid] - (BETA*dt/nodes[tid].mass)*dfs[tid];
}

/**
 * Computes the matrix-vector product Eu. All the pointer arguments are assumed to be
 * device pointers.
 *
 *      u:  device pointer to vector to multiply
 *    dFs:  device pointer to storage for per-particle dF matrices
 *    Aps:  device pointer to storage for per-particle Ap matrices
 * result:  device pointer to array store the values of Eu
 */
__host__ void computeEu( Particle *particles, int numParticles,
                         Grid *grid, ParticleGridNode *nodes, int numNodes,
                         float dt, vec3 *u, mat3 *dFs, mat3 *Aps, vec3 *dfs, vec3 *result )
{
    static const int threadCount = 128;

    computedF<<< numParticles/threadCount, threadCount >>>( particles, grid, dt, u, dFs );
    checkCudaErrors( hipDeviceSynchronize() );

    computeAp<<< numParticles/threadCount, threadCount >>>( particles, grid, dFs, Aps );
    checkCudaErrors( hipDeviceSynchronize() );

    dim3 blocks = dim3( numParticles/threadCount, 64 );
    dim3 threads = dim3( threadCount/64, 64 );
    computedf<<< blocks, threads >>>( particles, grid, Aps, dfs );
    checkCudaErrors( hipDeviceSynchronize() );

    computeResult<<< numNodes/threadCount, threadCount >>>( nodes, numNodes, dt, u, dfs, result );
    checkCudaErrors( hipDeviceSynchronize() );
}

__host__ void conjugateResidual( Particle *particles, int numParticles,
                                 Grid *grid, ParticleGridNode *nodes, int numNodes,
                                 float dt, vec3 *u, mat3 *dFs, mat3 *Aps, vec3 *dfs, vec3 *result )
{

}



#endif // IMPLICIT_H
