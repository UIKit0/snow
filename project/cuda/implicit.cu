#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   implicit.cu
**   Authors: evjang, mliberma, taparson, wyegelwe
**   Created: 26 Apr 2014
**
**************************************************************************/

#ifndef IMPLICIT_H
#define IMPLICIT_H

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define CUDA_INCLUDE
#include "geometry/grid.h"
#include "sim/material.h"
#include "sim/particle.h"
#include "sim/particlegridnode.h"
#include "cuda/vector.cu"

#include "cuda/atomic.cu"
#include "cuda/blas.cu"
#include "cuda/caches.h"
#include "cuda/decomposition.cu"
#include "cuda/weighting.cu"

#define BETA 0.5
#define MAX_ITERATIONS 30
#define STOPPING_EPSILON 1e-8

/**
 * Called over particles
 **/
#define VEC2IVEC( V ) ( glm::ivec3((int)V.x, (int)V.y, (int)V.z) )
__global__ void computedF(Particle *particles, Grid *grid, float dt, ParticleGridNode *nodes, vec3 *dus, Implicit::ParticleCache *pCaches ){
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particles[particleIdx];
    Implicit::ParticleCache &pCache = pCaches[particleIdx];
    mat3 dF(0.0f);

    const vec3 &pos = particle.position;
    const glm::ivec3 &dim = grid->dim;
    const float h = grid->h;

    // Compute neighborhood of particle in grid
    vec3 gridIndex = (pos - grid->pos) / h,
         gridMax = vec3::floor( gridIndex + vec3(2,2,2) ),
         gridMin = vec3::ceil( gridIndex - vec3(2,2,2) );
    glm::ivec3 maxIndex = glm::clamp( VEC2IVEC(gridMax), glm::ivec3(0,0,0), dim ),
               minIndex = glm::clamp( VEC2IVEC(gridMin), glm::ivec3(0,0,0), dim );

    mat3 vGradient(0.0f);

    // Fill dF
    int rowSize = dim.z+1;
    int pageSize = (dim.y+1)*rowSize;
    for ( int i = minIndex.x; i <= maxIndex.x; ++i ) {
        vec3 d, s;
        d.x = gridIndex.x - i;
        d.x *= ( s.x = ( d.x < 0 ) ? -1.f : 1.f );
        int pageOffset = i*pageSize;
        for ( int j = minIndex.y; j <= maxIndex.y; ++j ) {
            d.y = gridIndex.y - j;
            d.y *= ( s.y = ( d.y < 0 ) ? -1.f : 1.f );
            int rowOffset = pageOffset + j*rowSize;
            for ( int k = minIndex.z; k <= maxIndex.z; ++k ) {
                d.z = gridIndex.z - k;
                d.z *= ( s.z = ( d.z < 0 ) ? -1.f : 1.f );
                vec3 wg;
                weightGradient( -s, d, wg );

                vec3 du_j = dt * dus[rowOffset+k];
                dF += mat3::outerProduct(du_j, wg);

                vGradient += mat3::outerProduct(dt*nodes[rowOffset+k].velocity, wg);

            }
        }
    }

    pCache.dF = dF * particle.elasticF;

    pCache.FeHat = mat3::addIdentity(vGradient) * particle.elasticF;
    computePD( pCache.FeHat, pCache.ReHat, pCache.SeHat );
}

/** Currently computed in computedF, we could parallelize this and computedF but not sure what the time benefit would be*/
//__global__ void computeFeHat(Particle *particles, Grid *grid, float dt, ParticleGridNode *nodes, ACache *ACaches){
//    int particleIdx = blockIdx.x*blockDim.x + threadIdx.x;

//       Particle &particle = particles[particleIdx];
//       ACache &ACache = ACaches[particleIdx];

//       vec3 particleGridPos = (particle.position - grid->pos) / grid->h;
//       glm::ivec3 min = glm::ivec3(std::ceil(particleGridPos.x - 2), std::ceil(particleGridPos.y - 2), std::ceil(particleGridPos.z - 2));
//       glm::ivec3 max = glm::ivec3(std::floor(particleGridPos.x + 2), std::floor(particleGridPos.y + 2), std::floor(particleGridPos.z + 2));

//       mat3 vGradient(0.0f);

//       // Apply particles contribution of mass, velocity and force to surrounding nodes
//       min = glm::max(glm::ivec3(0.0f), min);
//       max = glm::min(grid->dim, max);
//       for (int i = min.x; i <= max.x; i++){
//           for (int j = min.y; j <= max.y; j++){
//               for (int k = min.z; k <= max.z; k++){
//                   int currIdx = grid->getGridIndex(i, j, k, grid->dim+1);
//                   ParticleGridNode &node = nodes[currIdx];

//                   vec3 wg;
//                   weightGradient(particleGridPos - vec3(i, j, k), wg);

//                   vGradient += mat3::outerProduct(dt*node.velocity, wg);
//               }
//           }
//       }

//       ACache.FeHat = mat3::addIdentity(vGradient) * particle.elasticF;
//       computePD(ACache.FeHat, ACache.ReHat, ACache.SeHat);
//}

__device__ void computedR(mat3 &dF, mat3 &Se, mat3 &Re, mat3 &dR){
    mat3 V = mat3::multiplyAtB(Re, dF) - mat3::multiplyAtB(dF, Re);

    // Solve for compontents of R^T * dR
    mat3 A = mat3(S[0]+S[4], S[5], -S[2], //remember, column major
                  S[5], S[0]+S[8], S[1],
                  -S[2], S[1], S[4]+S[8]);

    vec3 b(V[3], V[6], V[7]);
    vec3 x = mat3::solve(A, b);// Should replace this with a linear system solver function

    // Fill R^T * dR
    mat3 RTdR = mat3( 0, -x.x, -x.y, //remember, column major
                      x.x, 0, -x.z,
                      x.y, x.z, 0);

    dR = Re*RTdR;
}

/**
 * This function involves taking the partial derivative of the adjugate of F
 * with respect to each element of F. This process results in a 3x3 block matrix
 * where each block is the 3x3 partial derivative for an element of F
 *
 * Let F = [ a b c
 *           d e f
 *           g h i ]
 *
 * Let adjugate(F) = [ ei-hf  hc-bi  bf-ec
 *                     gf-di  ai-gc  dc-af
 *                     dh-ge  gb-ah  ae-db ]
 *
 * Then d/da (adjugate(F) = [ 0   0   0
 *                            0   i  -f
 *                            0  -h   e ]
 *
 * The other 8 partials will have similar form. See (and run) the code in
 * matlab/derivateAdjugateF.m for the full computation as well as to see where
 * these seemingly magic values came from.
 *
 *
 */
__device__ void compute_dJF_invTrans(mat3 &F, mat3 &dF, mat3 &dJF_invTrans){
    dJF_invTrans[0] = F[4]*dF[8] - F[5]*dF[5] + F[8]*dF[4] - F[7]*dF[7];
    dJF_invTrans[1] = F[5]*dF[2] - F[8]*dF[1] - F[3]*dF[8] + F[6]*dF[7];
    dJF_invTrans[2] = F[3]*dF[5] - F[4]*dF[2] + F[7]*dF[1] - F[6]*dF[4];
    dJF_invTrans[3] = F[2]*dF[5] - F[1]*dF[8] - F[8]*dF[3] + F[7]*dF[6];
    dJF_invTrans[4] = F[0]*dF[8] - F[2]*dF[2] + F[8]*dF[0] - F[6]*dF[6];
    dJF_invTrans[5] = F[1]*dF[2] - F[0]*dF[5] - F[7]*dF[0] + F[6]*dF[3];
    dJF_invTrans[6] = F[1]*dF[7] - F[2]*dF[4] + F[5]*dF[3] - F[4]*dF[6];
    dJF_invTrans[7] = F[2]*dF[1] - F[5]*dF[0] - F[0]*dF[7] + F[3]*dF[6];
    dJF_invTrans[8] = F[0]*dF[4] - F[1]*dF[1] + F[4]*dF[0] - F[3]*dF[3];
}

/**
 * Called over particles
 **/
// TODO: Replace JFe_invTrans with the trans of adjugate
__global__ void computeAp( Particle *particles, MaterialConstants *material, Implicit::ParticleCache *pCaches )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    Particle &particle = particles[particleIdx];
    Implicit::ParticleCache &pCache = pCaches[particleIdx];
    mat3 dF = pCache.dF;

    mat3 &Fp = particle.plasticF; //for the sake of making the code look like the math
    mat3 &Fe = pCache.FeHat;

    float Jpp = mat3::determinant(Fp);
    float Jep = mat3::determinant(Fe);

    float muFp = material->mu*__expf(material->xi*(1-Jpp));
    float lambdaFp = material->lambda*__expf(material->xi*(1-Jpp));

    mat3 &Re = pCache.ReHat;
    mat3 &Se = pCache.SeHat;

    mat3 dR;
    computedR(dF, Se, Re, dR);

    mat3 dJFe_invTrans;
    compute_dJF_invTrans(Fe, dF, dJFe_invTrans);

    mat3 jFe_invTrans = Jep * mat3::transpose(mat3::inverse(Fe));

    pCache.Ap = (2*muFp*(dF - dR) + lambdaFp*jFe_invTrans*mat3::innerProduct(jFe_invTrans, dF) + lambdaFp*(Jep - 1)*dJFe_invTrans);
}


__global__ void computedf( Particle *particles, Grid *grid, Implicit::ParticleCache *pCache, vec3 *df )
{
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    Particle &particle = particles[particleIdx];
    vec3 gridPos = (particle.position-grid->pos)/grid->h;

    glm::ivec3 ijk;
    Grid::gridIndexToIJK( threadIdx.y, glm::ivec3(4,4,4), ijk );
    ijk += glm::ivec3( gridPos.x-1, gridPos.y-1, gridPos.z-1 );

    if ( Grid::withinBoundsInclusive(ijk, glm::ivec3(0,0,0), grid->dim) ) {

        vec3 wg;
        vec3 nodePos(ijk);
        weightGradient( gridPos-nodePos, wg );
        vec3 df_j = -particle.volume * mat3::multiplyABt( pCache[particleIdx].Ap, particle.elasticF ) * wg;

        atomicAdd( &df[Grid::getGridIndex(ijk,grid->nodeDim())], df_j );
    }
}

__global__ void computeEuResult( ParticleGridNode *nodes, int numNodes, float dt, vec3 *u, vec3 *df, vec3 *result )
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if ( tid >= numNodes ) return;
    result[tid] = u[tid] - (BETA*dt/nodes[tid].mass)*df[tid];
}

/**
 * Computes the matrix-vector product Eu. All the pointer arguments are assumed to be
 * device pointers.
 */
__host__ void computeEu( const Particle *particles, int numParticles,
                         const Grid *grid, const ParticleGridNode *nodes, int numNodes,
                         float dt, const vec3 *u, const vec3 *df, vec3 *result, Implicit::ParticleCache *pCache )
{
    static const int threadCount = 128;

    dim3 blocks = dim3( numParticles/threadCount, 64 );
    dim3 threads = dim3( threadCount/64, 64 );

    computedf<<< blocks, threads >>>( particles, grid, pCache, df );
    checkCudaErrors( hipDeviceSynchronize() );
    computeEuResult<<< numNodes/threadCount, threadCount >>>( nodes, numNodes, dt, u, df, result );
    checkCudaErrors( hipDeviceSynchronize() );
}

__global__ void initializeVelocities( ParticleGridNode *nodes, int numNodes, vec3 *v )
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if ( tid >= numNodes ) return;
    v[tid] = nodes[tid].velocity;
}

__global__ void initializeRP( int numNodes, const vec3 *vstar, const vec3 *Ev0, vec3 *r0, vec3 *p0 )
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if ( tid >= numNodes ) return;
    r0[tid] = vstar[tid] - Ev0[tid];
    p0[tid] = r0[tid];
}

__global__ void initializeQ( int numNodes, vec3 *s0, vec3 *q0 )
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if ( tid >= numNodes ) return;
    q0[tid] = s0[tid];
}



__host__ void initializeConjugateResidual( Particle *particles, int numParticles,
                                           Grid *grid, ParticleGridNode *nodes, int numNodes,
                                           float dt, Implicit::NodeCache *nodeCache, float &gamma, float &alpha, Implicit::ParticleCache *pCache )
{
    static const int threadCount = 128;
    static const dim3 blocks( numNodes/threadCount );
    static const dim3 threads( threadCount );

    initializeVelocities<<< blocks, threads >>>( nodes, numNodes, nodeCache->v );
    checkCudaErrors( hipDeviceSynchronize() );

    computeEu( particles, numParticles, grid, nodes, numNodes, dt, nodeCache->v, nodeCache->df, nodeCache->r, pCache );
    initializeRP<<< blocks, threads >>>( numNodes, nodeCache->v, nodeCache->r, nodeCache->r, nodeCache->p );
    checkCudaErrors( hipDeviceSynchronize() );

    computeEu( particles, numParticles, grid, nodes, numNodes, dt, nodeCache->r, nodeCache->df, nodeCache->s, pCache );
    initializeQ<<< blocks, threads >>>( numNodes, nodeCache->s, nodeCache->q );
    checkCudaErrors( hipDeviceSynchronize() );

    gamma = innerProduct( numNodes, nodeCache->r, nodeCache->s, nodeCache->innerProduct );
    alpha = gamma / innerProduct( numNodes, nodeCache->q, nodeCache->q, nodeCache->innerProduct );
}

__global__ void finishConjugateResidualKernel( ParticleGridNode *nodes, int numNodes, const vec3 *v )
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if ( tid >= numNodes ) return;
    nodes[tid].velocity = v[tid];
}

__host__ void finishConjugateResidual( ParticleGridNode *nodes, int numNodes, const vec3 *v )
{
    finishConjugateResidualKernel<<< (numNodes+255)/256, 256 >>>( nodes, numNodes, v );
    hipDeviceSynchronize();
}

__host__ void computeNodeVelocitiesImplicit( Particle *particles, int numParticles,
                                             Grid *grid, ParticleGridNode *nodes, int numNodes,
                                             float dt, Implicit::NodeCache *nodeCache, Implicit::ParticleCache *pCache )
{
    float gamma, alpha, beta;
    initializeConjugateResidual( particles, numParticles, grid, nodes, numNodes, dt, nodeCache, gamma, alpha, pCache );

    int k = 0;
    float d;
    do {
        scaleAndAdd( numNodes, 1.f, nodeCache->v, alpha, nodeCache->p, nodeCache->v );
        scaleAndAdd( numNodes, 1.f, nodeCache->r, -alpha, nodeCache->q, nodeCache->r );
        computeEu( particles, numParticles, grid, nodes, numNodes, dt, nodeCache->r, nodeCache->df, nodeCache->s, pCache );
        beta = innerProduct( numNodes, nodeCache->r, nodeCache->s, nodeCache->innerProduct ) / gamma;
        gamma = beta * gamma;
        scaleAndAdd( numNodes, 1.f, nodeCache->r, beta, nodeCache->p, nodeCache->p );
        scaleAndAdd( numNodes, 1.f, nodeCache->s, beta, nodeCache->q, nodeCache->q );
        alpha = gamma / innerProduct( numNodes, nodeCache->q, nodeCache->q, nodeCache->innerProduct );
        d = alpha * alpha * innerProduct( numNodes, nodeCache->p, nodeCache->p, nodeCache->innerProduct ) / numNodes;
    } while ( k < MAX_ITERATIONS && d > STOPPING_EPSILON );

    finishConjugateResidual( nodes, numNodes, nodeCache->v );
}



#endif // IMPLICIT_H
