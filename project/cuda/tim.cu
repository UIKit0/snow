#include "hip/hip_runtime.h"
/**************************************************************************
**
**   SNOW - CS224 BROWN UNIVERSITY
**
**   tim.cu
**   Author: taparson
**   Created: 8 Apr 2014
**
**************************************************************************/

#ifndef TIM_CU
#define TIM_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <glm/geometric.hpp>
#include <glm/vec3.hpp>
#include <glm/mat3x3.hpp>

#define CUDA_INCLUDE
#include "sim/particle.h"
#include "geometry/grid.h"
#include "cuda/functions.h"

extern "C"  {
void groupParticlesTests();
void cumulativeSumTests();
void CSTest1();
void CSTest2();
void CSTest3();
void CSTest4();
void CSTest5();
void PGTest1();

}

__device__ int getGridIndex( int i, int j, int k, Grid* grid)  {
    glm::ivec3 dim = grid->dim;
    return (i*(dim.y*dim.z) + j*(dim.z) + k);
}

__global__ void rasterizeParticles( Particle *particleData, Grid *grid, int *particleToCell, int *cellParticleCount, int *particleOffsetInCell ) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    Particle p = particleData[index];
    glm::vec3 pos = p.position;
    pos-=grid->pos;
    pos/=grid->h;
    int gridIndex = getGridIndex((int)pos.x,(int)pos.y,(int)pos.z,grid);
    particleToCell[index] = gridIndex;
    particleOffsetInCell[index]=cellParticleCount[gridIndex]++;
}

__global__ void cumulativeSum(int *array, int M)  {
    int sum = 0;
    for(int i = 0; i < M; i++)  {
        sum+=array[i];
        array[i] = sum;
    }
}

__global__ void groupParticlesByCell( int *particleToCell, int *cellParticleIndex, int *particleOffsetInCell, int *gridParticles )  {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int gridIndex = particleToCell[index];
    int subPosition = particleOffsetInCell[index];
    int resultIndex = cellParticleIndex[gridIndex] + subPosition;
    gridParticles[resultIndex] = index;
}

void groupParticlesTests()  {
    printf("running particle grouping tests...\n");

    PGTest1();

    printf("done running particle grouping tests\n");
}

void PGTest1()  {
    int particleToCell[8] = {2,3,2,1,0,7,6,5};
    int cellParticleIndex[9] = {0,1,1,2,1,0,1,1,1};
    int particleOffsetInCell[8] = {0,0,1,0,0,0,0,0};
    int gridParticles[8] = {0,0,0,0,0,0,0,0};
    int *dev_particleToCell, *dev_cellParticleIndex, *dev_particleOffsetInCell, *dev_gridParticles;
    checkCudaErrors(hipMalloc((void**) &dev_particleToCell, 8*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_particleToCell,particleToCell,8*sizeof(int),hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**) &dev_cellParticleIndex, 9*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_cellParticleIndex,cellParticleIndex,9*sizeof(int),hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**) &dev_particleOffsetInCell, 8*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_particleOffsetInCell,particleOffsetInCell,8*sizeof(int),hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**) &dev_gridParticles, 8*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_gridParticles,gridParticles,8*sizeof(int),hipMemcpyHostToDevice));

    cumulativeSum<<<1,1>>>(dev_cellParticleIndex,9);
    hipDeviceSynchronize();
    groupParticlesByCell<<<8,1>>>(dev_particleToCell,dev_cellParticleIndex,dev_particleOffsetInCell,dev_gridParticles);

    hipDeviceSynchronize();
    hipMemcpy(gridParticles,dev_gridParticles,8*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(dev_particleToCell);
    hipFree(dev_cellParticleIndex);
    hipFree(dev_particleOffsetInCell);
    hipFree(dev_gridParticles);
    printf("{");
    for (int i=0; i < 8; i++)  {
        printf("%d,",gridParticles[i]);
    }
    printf("}\n");
}

void cumulativeSumTests()
{
    printf("running cumulative sum tests...\n");
    CSTest1();
    CSTest2();
    CSTest3();
    CSTest4();
    CSTest5();
    printf("done running cumulative sum tests\n");
}

void CSTest1()  {
    int array[5] = {0,1,2,3,4};
    int expected[5] = {0,1,3,6,10};
    printf("running test on array: [%d,%d,%d,%d,%d]...\n",array[0],array[1],array[2],array[3],array[4]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 5*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,5*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,5);
    hipMemcpy(array,dev_array,5*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 5; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d,%d,%d,%d,%d}",expected[0],expected[1],expected[2],expected[3],expected[4]);
            printf("    got: {%d,%d,%d,%d,%d}\n",array[0],array[1],array[2],array[3],array[4]);
            break;
        }
    }
}

void CSTest2()  {
    int array[5] = {5,1,2,3,4};
    int expected[5] = {5,6,8,11,15};
    printf("running test on array: [%d,%d,%d,%d,%d]...\n",array[0],array[1],array[2],array[3],array[4]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 5*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,5*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,5);
    hipMemcpy(array,dev_array,5*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 5; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d,%d,%d,%d,%d}",expected[0],expected[1],expected[2],expected[3],expected[4]);
            printf("    got: {%d,%d,%d,%d,%d}\n",array[0],array[1],array[2],array[3],array[4]);
            break;
        }
    }
}

void CSTest3()  {
    int array[1] = {5};
    int expected[1] = {5};
    printf("running test on array: [%d]...\n",array[0]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 1*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,1*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,1);
    hipMemcpy(array,dev_array,1*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 1; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d}",expected[0]);
            printf("    got: {%d}\n",array[0]);
            break;
        }
    }
}

void CSTest4()  {
    int array[1] = {0};
    int expected[1] = {0};
    printf("running test on array: [%d]...\n",array[0]);
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 1*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,1*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,1);
    hipMemcpy(array,dev_array,1*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 1; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {%d}",expected[0]);
            printf("    got: {%d}\n",array[0]);
            break;
        }
    }
}

void CSTest5()  {
    int array[0] = {};
    int expected[0] = {};
    printf("running test on array: []...\n");
    int *dev_array;
    checkCudaErrors(hipMalloc((void**) &dev_array, 0*sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_array,array,0*sizeof(int),hipMemcpyHostToDevice));
    cumulativeSum<<<1,1>>>(dev_array,0);
    hipMemcpy(array,dev_array,0*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dev_array);
    for (int i = 0; i < 0; i++)  {
        if (array[i] != expected[i])  {
            printf("failed test %d",1);
            printf("expected array: {}",expected[0]);
            printf("    got: {}\n");
            break;
        }
    }
}

#endif // TIM_CU

